#include "hip/hip_runtime.h"
#ifndef __DMD_CU__
#define __DMD_CU__

#define NVCC
#define ENABLE_CUDA
#define EPSILON 1e-12

#include <hip/hip_runtime.h>
#include "gpucell.cuh"
#include "indexer.h"
#include "gpubox.h"
#include "cu_functions.h"
#include <iostream>
#include <stdio.h>
#include "DelaunayMD.cuh"


//
//texture<int,1,hipReadModeElementType> dcc_tex;
//
__global__ void gpu_test_circumcenters_kernel(int *d_repair,
                                              int3 *d_circumcircles,
                                              float2 *d_pt,
                                              unsigned int *d_cell_sizes,
                                              int *d_cell_idx,
                                              int Nccs,
                                              int xsize,
                                              int ysize,
                                              float boxsize,
                                              gpubox Box,
                                              Index2D ci,
                                              Index2D cli,
                                              int *anyFail
                                              )
    {
    // read in the particle that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= Nccs)
        return;

    //the indices of particles forming the circumcircle
/*
    int i1 = d_circumcircles[3*idx];
    int i2 = d_circumcircles[3*idx+1];
    int i3 = d_circumcircles[3*idx+2];
*/
    int3 i1 = d_circumcircles[idx];
    //the vertex we will take to be the origin, and its cell position
    float2 v = d_pt[i1.x];
    float vz = 0.0;
    int ib=floorf(v.x/boxsize);
    int jb=floorf(v.y/boxsize);

    float2 p1real = d_pt[i1.y];
    float2 p2real = d_pt[i1.z];

    float2 pt1,pt2;
    Box.minDist(p1real,v,pt1);
    Box.minDist(p2real,v,pt2);

    //get the circumcircle
    float2 Q;
    float rad;
    Circumcircle(vz,vz,pt1.x,pt1.y,pt2.x,pt2.y,
                    Q.x,Q.y,rad);

    //look through cells for other particles
    bool badParticle = false;
    float2 ptnew,toCenter;
    int wcheck = ceilf(rad/boxsize)+1;
//
//    if (wcheck >= 3) badParticle = true;
//
    if(wcheck > xsize/2) wcheck = xsize/2;
    rad = rad*rad;
    for (int ii = -wcheck; ii <= wcheck; ++ii)
        {
        for (int jj = -wcheck; jj <= wcheck; ++jj)
            {
//if(badParticle) continue;
            int cx = (ib+ii);
            if(cx < 0) cx += xsize;
            if(cx >= xsize) cx -= xsize;
            int cy = (jb+jj);
            if(cy < 0) cy += ysize;
            if(cy >= ysize) cy -= ysize;

            int bin = ci(cx,cy);

            for (int pp = 0; pp < d_cell_sizes[bin]; ++pp)
                {
                int newidx = d_cell_idx[cli(pp,bin)];

                float2 pnreal = d_pt[newidx];
                Box.minDist(pnreal,v,ptnew);
                Box.minDist(ptnew,Q,toCenter);
                //if it's in the circumcircle, check that its not one of the three points
                if(toCenter.x*toCenter.x+toCenter.y*toCenter.y < rad)
                    {
                    if (newidx != i1.x && newidx != i1.y && newidx !=i1.z)
                        {
                        badParticle = true;
                        d_repair[newidx] = 1;
                        };
                    };

                };

            };
        };// end loop over cells

    if (badParticle)
        {
        *anyFail = 1;
        d_repair[i1.x] = 1;
        d_repair[i1.y] = 1;
        d_repair[i1.z] = 1;
        };

    return;
    };




__global__ void gpu_move_particles_kernel(float2 *d_points,
                                          float2 *d_disp,
                                          int N,
                                          gpubox Box
                                         )
    {
    // read in the particle that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    d_points[idx].x += d_disp[idx].x;
    d_points[idx].y += d_disp[idx].y;
    Box.putInBoxReal(d_points[idx]);
    return;
    };



bool gpu_test_circumcenters(int *d_repair,
                            int3 *d_ccs,
                            int Nccs,
                            float2 *d_pt,
                            unsigned int *d_cell_sizes,
                            int *d_idx,
                            int Np,
                            int xsize,
                            int ysize,
                            float boxsize,
                            gpubox &Box,
                            Index2D &ci,
                            Index2D &cli,
                            int &fail)
    {
    hipError_t code;
    unsigned int block_size = 128;
    if (Nccs < 128) block_size = 32;
    unsigned int nblocks  = Nccs/block_size + 1;
    
    fail = 0;
    int *anyFail;
    hipMalloc((void**)&anyFail,sizeof(int));
    hipMemcpy(anyFail,&fail,sizeof(int),hipMemcpyHostToDevice);

    code = hipGetLastError();
if(code!=hipSuccess)
    printf("testCircumcenters preliminary GPUassert: %s \n", hipGetErrorString(code));

    gpu_test_circumcenters_kernel<<<nblocks,block_size>>>(
                            d_repair,
                       //     d_redo2,
                            d_ccs,
                            d_pt,
                            d_cell_sizes,
                            d_idx,
                            Nccs,
                            xsize,
                            ysize,
                            boxsize,
                            Box,
                            ci,
                            cli,
                            anyFail
                            );
    hipMemcpy(&fail,anyFail,sizeof(int),hipMemcpyDeviceToHost);
    hipFree(anyFail);
    

//    cout << "Number of ccs to check: " << Nccs << endl;
//    hipBindTexture(0,dcc_tex,d_ccs,sizeof(int)*3*Nccs);
    /*
    bool *d_redo2;
    static const size_t size = Np*sizeof(bool);

    bool *bt = (bool*)malloc(Np*sizeof(bool));
    for (int nn = 0; nn < Np; ++nn) bt[nn]=false;

    code = hipMalloc((void **) &d_redo2,size);
if(code!=hipSuccess)
    printf("1 GPUassert: %s \n", hipGetErrorString(code));
    code = hipMemcpy(d_redo2,bt,size,hipMemcpyHostToDevice);
if(code!=hipSuccess)
    printf("2 GPUassert: %s \n", hipGetErrorString(code));

    code=hipDeviceSynchronize();
if(code!=hipSuccess)
    printf("3 GPUassert: %s \n", hipGetErrorString(code));

*/
    code = hipGetLastError();
if(code!=hipSuccess)
    printf("testCircumcenters GPUassert: %s \n", hipGetErrorString(code));

  //  hipUnbindTexture(dcc_tex);
/*
   code = hipMemcpy(bt,d_redo2,size,hipMemcpyDeviceToHost);

if(code!=hipSuccess)
    printf("4 GPUassert: %s \n", hipGetErrorString(code));

    hipFree(d_redo2);
    int totalwrong = 0;
    for (int nn = 0; nn < Np; ++nn)
        {
        if (bt[nn])
            totalwrong +=1;
//            cout << nn << "   " << bt[nn]<<endl;
        };
//    cout << endl;
    cout << "totalwrong = " << totalwrong << endl;

    free(bt);
*/
    return hipSuccess;
    };




bool gpu_move_particles(float2 *d_points,
                        float2 *d_disp,
                        int N,
                        gpubox &Box
                        )
    {
    hipError_t code;
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;

    gpu_move_particles_kernel<<<nblocks,block_size>>>(
                                                d_points,
                                                d_disp,
                                                N,
                                                Box
                                                );
    code = hipGetLastError();
if(code!=hipSuccess)
    printf("moveParticle GPUassert: %s \n", hipGetErrorString(code));

    return hipSuccess;
    };






#endif
