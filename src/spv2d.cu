#include "hip/hip_runtime.h"
#ifndef __SPV2D_CU__
#define __SPV2D_CU__

#define NVCC
#define ENABLE_CUDA
#define EPSILON 1e-12

#include <hip/hip_runtime.h>
#include "hiprand/hiprand_kernel.h"
#include "gpucell.cuh"
#include "spv2d.cuh"


#include "indexer.h"
#include "gpubox.h"
#include "cu_functions.h"
#include <iostream>
#include <stdio.h>

/*
__global__ void init_curand_kernel(unsigned long seed, hiprandState *state)
    {
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    hiprand_init(seed,idx,0,&state[idx]);
    return;
    };
*/

__global__ void gpu_compute_geometry_kernel(float2 *d_points,
                                          float2 *d_AP,
                                          float2 *d_voro,
                                          int *d_nn,
                                          int *d_n,
                                          int N,
                                          Index2D &n_idx,
                                          gpubox Box
                                        )
    {
    // read in the particle that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;

    float2 circumcenter, origin, nnextp, nlastp,pi,rij,rik;
    origin.x=0.0;origin.y=0.0;
    int neigh = d_nn[idx];

//printf(" id %i neighs %i\n",idx,neigh);

    pi = d_points[idx];
    nlastp = d_points[ d_n[n_idx(neigh-1,idx)] ];
    Box.minDist(nlastp,pi,rij);
//printf("if %i , (%f,%f), (%f,%f), (%f,%f)\n",idx,pi.x,pi.y,nlastp.x,nlastp.y,rij.x,rij.y);
    for (int nn = 0; nn < neigh; ++nn)
        {
        int nid = d_n[n_idx(nn,idx)];
        nnextp = d_points[ nid ];
        Box.minDist(nnextp,pi,rik);
//printf("%i %i  gpu (%f,%f), (%f,%f), (%f,%f)\n",nn,nid,origin.x,origin.y,rij.x,rij.y,rik.x,rik.y);
        Circumcenter(origin,rij,rik,circumcenter);
        d_voro[n_idx(nn,idx)] = circumcenter;
        rij=rik;
        };

    //think about how to incorporate this into the loop above
    float Varea = 0.0;
    float Vperi= 0.0;
    float2 vlast,vnext;
    vlast = d_voro[n_idx(neigh-1,idx)];
    for (int nn = 0; nn < neigh; ++nn)
        {
        vnext = d_voro[n_idx(nn,idx)];
        Varea += TriangleArea(vlast,vnext);
        float dx = vlast.x - vnext.x;
        float dy = vlast.y - vnext.y;
        Vperi += sqrt(dx*dx+dy*dy);
if (idx  == 40 ) printf("(%f,%f)\t\n",vlast.x,vlast.y);
        vlast=vnext;
        };

    d_AP[idx].x=Varea;
    d_AP[idx].y=Vperi;
//printf("last voro: (%f,%f); area,peri (%f,%f)\t\n",vnext.x,vnext.y,Varea,Vperi);

    return;
    };



__global__ void gpu_displace_and_rotate_kernel(float2 *d_points,
                                          float2 *d_force,
                                          float *d_directors,
                                          int N,
                                          float dt,
                                          float Dr,
                                          float v0,
                                          int seed,
//                                          hiprandState *states,
                                          gpubox Box
                                         )
    {
    // read in the particle that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;

    hiprandState_t randState;
    hiprand_init(seed,//seed first
                0,   // sequence -- only important for multiple cores
                0,   //offset. advance by sequence by 1 plus this value
                &randState);

    float dirx = cosf(d_directors[idx]);
    float diry = sinf(d_directors[idx]);
    //float angleDiff = hiprand_normal(&states[idx])*sqrt(2.0*dt*Dr);
    float angleDiff = hiprand_normal(&randState)*sqrt(2.0*dt*Dr);
    d_directors[idx] += angleDiff;

 //   float dx = dt*(v0*dirx + d_force[idx].x);
//if (idx == 0) printf("x-displacement = %e\n",dx);
    d_points[idx].x += dt*(v0*dirx + d_force[idx].x);
    d_points[idx].y += dt*(v0*diry + d_force[idx].y);
    Box.putInBoxReal(d_points[idx]);
    return;
    };


//////////////
//kernel callers
//



/*
bool gpu_init_curand(hiprandState *states,
                    unsigned long seed,
                    int N)
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;


    hipMalloc((void **)&states,nblocks*block_size*sizeof(hiprandState) );
    init_curand_kernel<<<nblocks,block_size>>>(seed,states);
    return hipSuccess;
    };
*/


bool gpu_compute_geometry(float2 *d_points,
                        float2   *d_AP,
                        float2   *d_voro,
                        int      *d_nn,
                        int      *d_n,
                        int      N,
                        Index2D  &n_idx,
                        gpubox &Box
                        )
    {
    hipError_t code;
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;

    gpu_compute_geometry_kernel<<<nblocks,block_size>>>(
                                                d_points,
                                                d_AP,
                                                d_voro,
                                                d_nn,
                                                d_n,
                                                N,
                                                n_idx,
                                                Box
                                                );

    code = hipGetLastError();
    if(code!=hipSuccess)
    printf("compute geometry GPUassert: %s \n", hipGetErrorString(code));

    return hipSuccess;
    };


bool gpu_displace_and_rotate(float2 *d_points,
                        float2 *d_force,
                        float  *d_directors,
                        int N,
                        float dt,
                        float Dr,
                        float v0,
                        int seed,
  //                      hiprandState *states,
                        gpubox &Box
                        )
    {
    hipError_t code;
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;

    gpu_displace_and_rotate_kernel<<<nblocks,block_size>>>(
                                                d_points,
                                                d_force,
                                                d_directors,
                                                N,
                                                dt,
                                                Dr,
                                                v0,
                                                seed,
    //                                            states,
                                                Box
                                                );
    code = hipGetLastError();
    if(code!=hipSuccess)
    printf("displaceAndRotate GPUassert: %s \n", hipGetErrorString(code));

    return hipSuccess;
    };


#endif
