#include "hip/hip_runtime.h"
#define NVCC
#define ENABLE_CUDA

#include <hip/hip_runtime.h>
#include "hiprand/hiprand_kernel.h"
#include "cellListGPU.cuh"
#include "spv2d.cuh"

#include "indexer.h"
#include "gpubox.h"
#include "cu_functions.h"
#include <iostream>
#include <stdio.h>
#include "Matrix.h"

/*!
A file defining some global kernels for use in the spv2d class
*/

/*!
    \addtogroup spvKernels
    @{
*/

/*! 
  Each thread -- corresponding to each Voronoi cell -- is initialized with a different sequence
  of the same seed of a cudaRNG
*/
__global__ void init_curand_kernel(hiprandState *state, int N,int Timestep,int GlobalSeed)
    {
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx >=N)
        return;

    hiprand_init(GlobalSeed,idx,Timestep,&state[idx]);
    return;
    };


/*!
  Each cell has a force contribution due to the derivative of the energy with respect to each of
  its voronoi vertices... add them up to get the force per cell.
  */
__global__ void gpu_sum_forces_kernel(const Dscalar2* __restrict__ d_forceSets,
                                      Dscalar2* __restrict__ d_forces,
                                      const int* __restrict__      d_nn,
                                      int     N,
                                      Index2D n_idx
                                     )
    {
    // read in the particle that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;

    int neigh = d_nn[idx];
    Dscalar2 temp;
    temp.x=0.0;temp.y=0.0;
    for (int nn = 0; nn < neigh; ++nn)
        {
        Dscalar2 val = d_forceSets[n_idx(nn,idx)];
        temp.x+=val.x;
        temp.y+=val.y;
        };

    d_forces[idx]=temp;

    };

/*!
  add up force sets, as above, but keep track of exclusions
  */
__global__ void gpu_sum_forces_with_exclusions_kernel(const Dscalar2* __restrict__ d_forceSets,
                                      Dscalar2* __restrict__ d_forces,
                                      Dscalar2* __restrict__ d_external_forces,
                                      const int* __restrict__ d_exes,
                                      const int* __restrict__ d_nn,
                                      int     N,
                                      Index2D n_idx
                                     )
    {
    // read in the particle that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;

    int neigh = d_nn[idx];
    Dscalar2 temp;
    temp.x=0.0;temp.y=0.0;
    for (int nn = 0; nn < neigh; ++nn)
        {
        Dscalar2 val = d_forceSets[n_idx(nn,idx)];
        temp.x+=val.x;
        temp.y+=val.y;
        };
    if (d_exes[idx] ==0)
        {
        d_forces[idx]=temp;
        d_external_forces[idx] = make_Dscalar2(0.0,0.0);
        }
    else
        {
        d_forces[idx]=make_Dscalar2(0.0,0.0);
        d_external_forces[idx] = make_Dscalar2(-temp.x,-temp.y);
        };

    };

/*!
  the force on a particle is decomposable into the force contribution from each of its voronoi
  vertices...calculate those sets of forces
  */
__global__ void gpu_force_sets_kernel(const Dscalar2* __restrict__ d_points,
                                      const Dscalar2* __restrict__ d_AP,
                                      const Dscalar2*  __restrict__ d_APpref,
                                      const int2* __restrict__ d_delSets,
                                      const int* __restrict__ d_delOther,
                                      const Dscalar2* __restrict__ d_vc,
                                      const Dscalar4* __restrict__ d_vln,
                                      Dscalar2* __restrict__ d_forceSets,
                                      const int2* __restrict__ d_nidx,
                                      Dscalar   KA,
                                      Dscalar   KP,
                                      int     computations,
                                      Index2D n_idx,
                                      gpubox Box
                                     )
    {
    unsigned int tidx = blockDim.x * blockIdx.x + threadIdx.x;
    if (tidx >= computations)
        return;

    //which particle are we evaluating, and which neighbor
    int pidx = d_nidx[tidx].x;
    int nn = d_nidx[tidx].y;
    int nidx=n_idx(nn,pidx);

    //local variables declared...
    Dscalar2 dAdv,dPdv;
    Dscalar2 dEdv;
    Dscalar  Adiff, Pdiff;
    Dscalar2 dlast, dnext,dcl,dnc;
    Dscalar  dlnorm,dnnorm,dclnorm,dncnorm;
    Dscalar2 vlast,vcur,vnext,vother;

    //logically, I want these variables:
    //Dscalar2 pi, rij, rik,pno;
    //they will simply re-use
    //     dlast, dnext, dcl, dnc, respectively
    //to reduce register usage


    //Great...access the Delaunay neighbors and the relevant other point
    int2 neighs;
    dlast   = d_points[pidx];

    neighs = d_delSets[nidx];

    Box.minDist(d_points[neighs.x],dlast,dnext);
    Box.minDist(d_points[neighs.y],dlast,dcl);
    Box.minDist(d_points[d_delOther[nidx]],dlast,dnc);

    //first, compute the derivative of the main voro point w/r/t pidx's position
    Matrix2x2 dhdr;
    getdhdr(dhdr,dnext,dcl);

    //finally, compute all of the forces
    //pnm1 is rij (dnext), pn1 is rik
    vcur = d_vc[nidx];
    Dscalar4 vvv = d_vln[nidx];
    vlast.x = vvv.x; vlast.y = vvv.y;
    vnext.x = vvv.z; vnext.y = vvv.w;

    Circumcenter(dnext,dcl,dnc,vother);


    //self terms
    dAdv.x = 0.5*(vlast.y-vnext.y);
    dAdv.y = 0.5*(vnext.x-vlast.x);
    dlast.x = vlast.x-vcur.x;
    dlast.y=vlast.y-vcur.y;
    dlnorm = sqrt(dlast.x*dlast.x+dlast.y*dlast.y);
    dnext.x = vcur.x-vnext.x;
    dnext.y = vcur.y-vnext.y;
    dnnorm = sqrt(dnext.x*dnext.x+dnext.y*dnext.y);
#ifdef SCALARFLOAT
    if(dnnorm < THRESHOLD)
        dnnorm = THRESHOLD;
    if(dlnorm < THRESHOLD)
        dlnorm = THRESHOLD;
#endif
    //save a few of these differences for later...
    //dcl.x = -dlast.x;dcl.y = -dlast.y;
    //dnc.x=-dnext.x;dnc.y=-dnext.y;
    dcl.x = dlast.x; dcl.y = dlast.y;
    dnc.x = dnext.x; dnc.y = dnext.y;
    dclnorm=dlnorm;
    dncnorm=dnnorm;

    dPdv.x = dlast.x/dlnorm - dnext.x/dnnorm;
    dPdv.y = dlast.y/dlnorm - dnext.y/dnnorm;
    Adiff = KA*(d_AP[pidx].x - d_APpref[pidx].x);
    Pdiff = KP*(d_AP[pidx].y - d_APpref[pidx].y);

    //replace all "multiply-by-two's" with a single one at the end...saves 10 mult operations
    dEdv.x  = Adiff*dAdv.x + Pdiff*dPdv.x;
    dEdv.y  = Adiff*dAdv.y + Pdiff*dPdv.y;

    //other terms...k first...
    dAdv.x = 0.5*(vnext.y-vother.y);
    dAdv.y = 0.5*(vother.x-vnext.x);
    dnext.x = vcur.x-vother.x;
    dnext.y = vcur.y-vother.y;
    dnnorm = sqrt(dnext.x*dnext.x+dnext.y*dnext.y);
#ifdef SCALARFLOAT
    if(dnnorm < THRESHOLD)
        dnnorm = THRESHOLD;
#endif
    dPdv.x = -dnc.x/dncnorm - dnext.x/dnnorm;
    dPdv.y = -dnc.y/dncnorm - dnext.y/dnnorm;
    Adiff = KA*(d_AP[neighs.y].x - d_APpref[neighs.y].x);
    Pdiff = KP*(d_AP[neighs.y].y - d_APpref[neighs.y].y);

    dEdv.x  += Adiff*dAdv.x + Pdiff*dPdv.x;
    dEdv.y  += Adiff*dAdv.y + Pdiff*dPdv.y;

    //...and then j
    dAdv.x = 0.5*(vother.y-vlast.y);
    dAdv.y = 0.5*(vlast.x-vother.x);
    //dlast is now -(dnext) from the K calculation
    //dlast.x = -dnext.x;
    //dlast.y = -dnext.y;
    //dlnorm = dnnorm;
    dPdv.x = -dnext.x/dnnorm + dcl.x/dclnorm;
    dPdv.y = -dnext.y/dnnorm + dcl.y/dclnorm;
    Adiff = KA*(d_AP[neighs.x].x - d_APpref[neighs.x].x);
    Pdiff = KP*(d_AP[neighs.x].y - d_APpref[neighs.x].y);

    dEdv.x  += Adiff*dAdv.x + Pdiff*dPdv.x;
    dEdv.y  += Adiff*dAdv.y + Pdiff*dPdv.y;

    dEdv.x *= 2.0;
    dEdv.y *= 2.0;

    d_forceSets[nidx] = dEdv*dhdr;

    return;
    };

/*!
The force on a particle is decomposable into the force contribution from each of its voronoi vertices
...calculate those sets of forces with an additional tension term between cells of different type
*/
__global__ void gpu_force_sets_tensions_kernel(const Dscalar2* __restrict__ d_points,
                                          const Dscalar2* __restrict__ d_AP,
                                          const Dscalar2* __restrict__ d_APpref,
                                          const int2* __restrict__ d_delSets,
                                          const int* __restrict__ d_delOther,
                                          const Dscalar2* __restrict__ d_vc,
                                          const Dscalar4* __restrict__ d_vln,
                                          Dscalar2* __restrict__ d_forceSets,
                                          const int2* __restrict__ d_nidx,
                                          const int* __restrict__ d_cellTypes,
                                          Dscalar   KA,
                                          Dscalar   KP,
                                          Dscalar   gamma,
                                          int     computations,
                                          Index2D n_idx,
                                          gpubox Box
                                        )
    {
    unsigned int tidx = blockDim.x * blockIdx.x + threadIdx.x;
    if (tidx >= computations)
        return;

    //which particle are we evaluating, and which neighbor
    int pidx = d_nidx[tidx].x;
    int nn = d_nidx[tidx].y;
    int nidx=n_idx(nn,pidx);

    //Great...access the Delaunay neighbors and the relevant other point
    Dscalar2 pi   = d_points[pidx];

    int2 neighs = d_delSets[nidx];
    int neighOther = d_delOther[nidx];
    Dscalar2 rij, rik,pno;

    Box.minDist(d_points[neighs.x],pi,rij);
    Box.minDist(d_points[neighs.y],pi,rik);
    Box.minDist(d_points[neighOther],pi,pno);

    //first, compute the derivative of the main voro point w/r/t pidx's position
    Matrix2x2 dhdr;
    getdhdr(dhdr,rij,rik);

    //finally, compute all of the forces
    //pnm1 is rij, pn1 is rik
    Dscalar2 vlast,vcur,vnext,vother;
    vcur = d_vc[nidx];
    Dscalar4 vvv = d_vln[nidx];
    vlast.x = vvv.x; vlast.y = vvv.y;
    vnext.x = vvv.z; vnext.y = vvv.w;
    Circumcenter(rij,rik,pno,vother);


    Dscalar2 dAdv,dPdv,dTdv;
    Dscalar2 dEdv;
    Dscalar  Adiff, Pdiff;
    Dscalar2 dlast, dnext,dcl,dnc;
    Dscalar  dlnorm,dnnorm,dclnorm,dncnorm;
    bool Tik = false;
    bool Tij = false;
    bool Tjk = false;
    if (d_cellTypes[pidx] != d_cellTypes[neighs.y]) Tik = true;
    if (d_cellTypes[pidx] != d_cellTypes[neighs.x]) Tij = true;
    if (d_cellTypes[neighs.y] != d_cellTypes[neighs.x]) Tjk = true;
    //neighs.y is "baseNeigh" of cpu routing... neighs.x is "otherNeigh"....neighOther is "DT_other_idx"

    //self terms
    dAdv.x = 0.5*(vlast.y-vnext.y);
    dAdv.y = 0.5*(vnext.x-vlast.x);
    dlast.x = vlast.x-vcur.x;
    dlast.y=vlast.y-vcur.y;
    dlnorm = sqrt(dlast.x*dlast.x+dlast.y*dlast.y);
    dnext.x = vcur.x-vnext.x;
    dnext.y = vcur.y-vnext.y;
    dnnorm = sqrt(dnext.x*dnext.x+dnext.y*dnext.y);
    if(dnnorm < THRESHOLD)
        dnnorm = THRESHOLD;
    if(dlnorm < THRESHOLD)
        dlnorm = THRESHOLD;

    //save a few of these differences for later...
    dcl.x = -dlast.x;dcl.y = -dlast.y;
    dnc.x=-dnext.x;dnc.y=-dnext.y;
    dclnorm=dlnorm;
    dncnorm=dnnorm;

    dPdv.x = dlast.x/dlnorm - dnext.x/dnnorm;
    dPdv.y = dlast.y/dlnorm - dnext.y/dnnorm;
    dTdv.x = 0.0; dTdv.y = 0.0;
    if(Tik)
        {
        dTdv.x -= dnext.x/dnnorm;
        dTdv.y -= dnext.y/dnnorm;
        };
    if(Tij)
        {
        dTdv.x += dlast.x/dlnorm;
        dTdv.y += dlast.y/dlnorm;
        };

    Adiff = KA*(d_AP[pidx].x - d_APpref[pidx].x);
    Pdiff = KP*(d_AP[pidx].y - d_APpref[pidx].y);

    //defer a global factor of two to the very end...saves six multiplications...
    dEdv.x  =  Adiff*dAdv.x + Pdiff*dPdv.x + 0.5*gamma*dTdv.x;
    dEdv.y  =  Adiff*dAdv.y + Pdiff*dPdv.y + 0.5*gamma*dTdv.y;

    //other terms...k first...
    dAdv.x = 0.5*(vnext.y-vother.y);
    dAdv.y = 0.5*(vother.x-vnext.x);
    dnext.x = vcur.x-vother.x;
    dnext.y = vcur.y-vother.y;
    dnnorm = sqrt(dnext.x*dnext.x+dnext.y*dnext.y);
    if(dnnorm < THRESHOLD)
        dnnorm = THRESHOLD;
    dPdv.x = dnc.x/dncnorm - dnext.x/dnnorm;
    dPdv.y = dnc.y/dncnorm - dnext.y/dnnorm;
    Adiff = KA*(d_AP[neighs.y].x - d_APpref[neighs.y].x);
    Pdiff = KP*(d_AP[neighs.y].y - d_APpref[neighs.y].y);
    dTdv.x = 0.0; dTdv.y = 0.0;
    if(Tik)
        {
        dTdv.x += dnc.x/dncnorm;
        dTdv.y += dnc.y/dncnorm;
        };
    if(Tjk)
        {
        dTdv.x -= dnext.x/dnnorm;
        dTdv.y -= dnext.y/dnnorm;
        };

    dEdv.x  += Adiff*dAdv.x + Pdiff*dPdv.x + 0.5*gamma*dTdv.x;
    dEdv.y  += Adiff*dAdv.y + Pdiff*dPdv.y + 0.5*gamma*dTdv.y;

    //...and then j
    dAdv.x = 0.5*(vother.y-vlast.y);
    dAdv.y = 0.5*(vlast.x-vother.x);
    dlast.x = -dnext.x;
    dlast.y = -dnext.y;
    dlnorm = dnnorm;
    dPdv.x = dlast.x/dlnorm - dcl.x/dclnorm;
    dPdv.y = dlast.y/dlnorm - dcl.y/dclnorm;
    Adiff = KA*(d_AP[neighs.x].x - d_APpref[neighs.x].x);
    Pdiff = KP*(d_AP[neighs.x].y - d_APpref[neighs.x].y);
    dTdv.x = 0.0; dTdv.y = 0.0;
    if(Tij)
        {
        dTdv.x -= dcl.x/dclnorm;
        dTdv.y -= dcl.y/dclnorm;
        };
    if(Tjk)
        {
        dTdv.x += dlast.x/dlnorm;
        dTdv.y += dlast.y/dlnorm;
        };

    dEdv.x  +=  Adiff*dAdv.x + Pdiff*dPdv.x + 0.5*gamma*dTdv.x;
    dEdv.y  +=  Adiff*dAdv.y + Pdiff*dPdv.y + 0.5*gamma*dTdv.y;

    dEdv.x *= 2.0;
    dEdv.y *= 2.0;

    d_forceSets[nidx] = dEdv*dhdr;

    return;
    };



/*!
  Since the cells are guaranteed to be convex, the area of the cell is the sum of the areas of
  the triangles formed by consecutive Voronoi vertices
  */
__global__ void gpu_compute_geometry_kernel(const Dscalar2* __restrict__ d_points,
                                          Dscalar2* __restrict__ d_AP,
                                          const int* __restrict__ d_nn,
                                          const int* __restrict__ d_n,
                                          Dscalar2* __restrict__ d_vc,
                                          Dscalar4* __restrict__ d_vln,
                                          int N,
                                          Index2D n_idx,
                                          gpubox Box
                                        )
    {
    // read in the particle that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;

    Dscalar2  nnextp, nlastp,pi,rij,rik,vlast,vnext,vfirst;

    int neigh = d_nn[idx];
    Dscalar Varea = 0.0;
    Dscalar Vperi= 0.0;

    pi = d_points[idx];
    nlastp = d_points[ d_n[n_idx(neigh-1,idx)] ];
    nnextp = d_points[ d_n[n_idx(0,idx)] ];

    Box.minDist(nlastp,pi,rij);
    Box.minDist(nnextp,pi,rik);
    Circumcenter(rij,rik,vfirst);
    vlast = vfirst;

    //set the VoroCur to this voronoi vertex
    //the convention is that nn=0 in this routine should be nn = 1 in the force sets calculation
    d_vc[n_idx(1,idx)] = vlast;

    for (int nn = 1; nn < neigh; ++nn)
        {
        rij = rik;
        int nid = d_n[n_idx(nn,idx)];
        nnextp = d_points[ nid ];
        Box.minDist(nnextp,pi,rik);
        Circumcenter(rij,rik,vnext);

        //fill in the VoroCur structure

        int idc = n_idx(nn+1,idx);
        if(nn == neigh-1)
            idc = n_idx(0,idx);

        d_vc[idc]=vnext;

        //...and back to computing the geometry
        Varea += TriangleArea(vlast,vnext);
        Dscalar dx = vlast.x - vnext.x;
        Dscalar dy = vlast.y - vnext.y;
        Vperi += sqrt(dx*dx+dy*dy);
        vlast=vnext;
        };
    Varea += TriangleArea(vlast,vfirst);
    Dscalar dx = vlast.x - vfirst.x;
    Dscalar dy = vlast.y - vfirst.y;
    Vperi += sqrt(dx*dx+dy*dy);

    //it's more memory-access friendly to now fill in the VoroLastNext structure separately
    vlast = d_vc[n_idx(neigh-1,idx)];
    vfirst = d_vc[n_idx(0,idx)];
    for (int nn = 0; nn < neigh; ++nn)
        {
        int idn = n_idx(nn+1,idx);
        if(nn == neigh-1) idn = n_idx(0,idx);
        vnext = d_vc[idn];

        int idc = n_idx(nn,idx);
        d_vln[idc].x = vlast.x;
        d_vln[idc].y = vlast.y;
        d_vln[idc].z = vnext.x;
        d_vln[idc].w = vnext.y;

        vlast = vfirst;
        vfirst = vnext;
        };

    d_AP[idx].x=Varea;
    d_AP[idx].y=Vperi;

    return;
    };


/*!
  move particles according to their motility and forces...each thread checks out its RNG, advances
  it by one increment, and returns it.
  */
__global__ void gpu_displace_and_rotate_kernel(Dscalar2 *d_points,
                                          Dscalar2 *d_force,
                                          Dscalar *d_directors,
                                          Dscalar2 *d_motility,
                                          int N,
                                          Dscalar dt,
                                          int seed,
                                          hiprandState *states,
                                          gpubox Box
                                         )
    {
    // read in the particle that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;

    hiprandState_t randState;

    randState=states[idx];
    Dscalar dirx = Cos(d_directors[idx]);
    Dscalar diry = Sin(d_directors[idx]);
    Dscalar v0 = d_motility[idx].x;
    Dscalar Dr = d_motility[idx].y;
    Dscalar angleDiff = cur_norm(&randState)*sqrt(2.0*dt*Dr);
    d_directors[idx] += angleDiff;

    states[idx] = randState;

    d_points[idx].x += dt*(v0*dirx + d_force[idx].x);
    d_points[idx].y += dt*(v0*diry + d_force[idx].y);
    Box.putInBoxReal(d_points[idx]);
    return;
    };

////////////////
//kernel callers
////////////////

//!Call the kernel to initialize a different RNG for each particle
bool gpu_init_curand(hiprandState *states,
                    int N,
                    int Timestep,
                    int GlobalSeed)
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;


    init_curand_kernel<<<nblocks,block_size>>>(states,N,Timestep,GlobalSeed);
    HANDLE_ERROR(hipGetLastError());
    //hipDeviceSynchronize();
    return hipSuccess;
    };

//!Call the kernel to compute the geometry
bool gpu_compute_geometry(Dscalar2 *d_points,
                        Dscalar2   *d_AP,
                        int      *d_nn,
                        int      *d_n,
                        Dscalar2 *d_vc,
                        Dscalar4 *d_vln,
                        int      N,
                        Index2D  &n_idx,
                        gpubox &Box
                        )
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;

    gpu_compute_geometry_kernel<<<nblocks,block_size>>>(
                                                d_points,
                                                d_AP,
                                                d_nn,
                                                d_n,
                                                d_vc,
                                                d_vln,
                                                N,
                                                n_idx,
                                                Box
                                                );
    //hipDeviceSynchronize();
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

//!Call the kernel to move particles around
bool gpu_displace_and_rotate(Dscalar2 *d_points,
                        Dscalar2 *d_force,
                        Dscalar  *d_directors,
                        Dscalar2 *d_motility,
                        int N,
                        Dscalar dt,
                        int timestep,
                        hiprandState *states,
                        gpubox &Box
                        )
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;

    gpu_displace_and_rotate_kernel<<<nblocks,block_size>>>(
                                                d_points,
                                                d_force,
                                                d_directors,
                                                d_motility,
                                                N,
                                                dt,
                                                timestep,
                                                states,
                                                Box
                                                );
    //hipDeviceSynchronize();
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

//!Call the kernel to compute the force sets
bool gpu_force_sets(Dscalar2 *d_points,
                    Dscalar2 *d_AP,
                    Dscalar2 *d_APpref,
                    int2   *d_delSets,
                    int    *d_delOther,
                    Dscalar2 *d_vc,
                    Dscalar4 *d_vln,
                    Dscalar2 *d_forceSets,
                    int2   *d_nidx,
                    Dscalar  KA,
                    Dscalar  KP,
                    int    NeighIdxNum,
                    Index2D &n_idx,
                    gpubox &Box
                    )
    {
    unsigned int block_size = 128;
    if (NeighIdxNum < 128) block_size = 32;
    unsigned int nblocks  = NeighIdxNum/block_size + 1;

    gpu_force_sets_kernel<<<nblocks,block_size>>>(
                                                d_points,
                                                d_AP,
                                                d_APpref,
                                                d_delSets,
                                                d_delOther,
                                                d_vc,
                                                d_vln,
                                                d_forceSets,
                                                d_nidx,
                                                KA,
                                                KP,
                                                NeighIdxNum,
                                                n_idx,
                                                Box
                                                );
    HANDLE_ERROR(hipGetLastError());
    //hipDeviceSynchronize();
    return hipSuccess;
    };



//!Call the kernel to compute force sets with additional tension terms
bool gpu_force_sets_tensions(Dscalar2 *d_points,
                    Dscalar2 *d_AP,
                    Dscalar2 *d_APpref,
                    int2   *d_delSets,
                    int    *d_delOther,
                    Dscalar2 *d_vc,
                    Dscalar4 *d_vln,
                    Dscalar2 *d_forceSets,
                    int2   *d_nidx,
                    int    *d_cellTypes,
                    Dscalar  KA,
                    Dscalar  KP,
                    Dscalar  gamma,
                    int    NeighIdxNum,
                    Index2D &n_idx,
                    gpubox &Box
                    )
    {
    unsigned int block_size = 128;
    if (NeighIdxNum < 128) block_size = 32;
    unsigned int nblocks  = NeighIdxNum/block_size + 1;

    gpu_force_sets_tensions_kernel<<<nblocks,block_size>>>(
                                                d_points,
                                                d_AP,
                                                d_APpref,
                                                d_delSets,
                                                d_delOther,
                                                d_vc,
                                                d_vln,
                                                d_forceSets,
                                                d_nidx,
                                                d_cellTypes,
                                                KA,
                                                KP,
                                                gamma,
                                                NeighIdxNum,
                                                n_idx,
                                                Box
                                                );
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

//!call the kernel to add up the forces
bool gpu_sum_force_sets(
                        Dscalar2 *d_forceSets,
                        Dscalar2 *d_forces,
                        int    *d_nn,
                        int     N,
                        Index2D &n_idx
                        )
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;

    gpu_sum_forces_kernel<<<nblocks,block_size>>>(
                                            d_forceSets,
                                            d_forces,
                                            d_nn,
                                            N,
                                            n_idx
            );
    //hipDeviceSynchronize();
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };


//!call the kernel to add up forces with particle exclusions
bool gpu_sum_force_sets_with_exclusions(
                        Dscalar2 *d_forceSets,
                        Dscalar2 *d_forces,
                        Dscalar2 *d_external_forces,
                        int    *d_exes,
                        int    *d_nn,
                        int     N,
                        Index2D &n_idx
                        )
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;

    gpu_sum_forces_with_exclusions_kernel<<<nblocks,block_size>>>(
                                            d_forceSets,
                                            d_forces,
                                            d_external_forces,
                                            d_exes,
                                            d_nn,
                                            N,
                                            n_idx
            );
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

/** @} */ //end of group declaration
