#include "hip/hip_runtime.h"
#ifndef __SPV2D_CU__
#define __SPV2D_CU__

#define NVCC
#define ENABLE_CUDA
#define EPSILON 1e-12
#define THRESHOLD 1e-8

#include <hip/hip_runtime.h>
#include "hiprand/hiprand_kernel.h"
#include "gpucell.cuh"
#include "spv2d.cuh"

#include "indexer.h"
#include "gpubox.h"
#include "cu_functions.h"
#include <iostream>
#include <stdio.h>
#include "Matrix.h"

/*
__global__ void init_curand_kernel(unsigned long seed, hiprandState *state)
    {
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    hiprand_init(seed,idx,0,&state[idx]);
    return;
    };
*/

__global__ void gpu_sum_forces_kernel(float2 *d_forceSets,
                                      float2 *d_forces,
                                      int    *d_nn,
                                      int     N,
                                      Index2D n_idx
                                     )
    {
    // read in the particle that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    
    int neigh = d_nn[idx];
    float2 temp;
    temp.x=0.0;temp.y=0.0;
    for (int nn = 0; nn < neigh; ++nn)
        {
        float2 val = d_forceSets[n_idx(nn,idx)];
        temp.x+=val.x;
        temp.y+=val.y;
        };
//    if(!::isfinite(temp.x)) temp.x = 0.;
//    if(!::isfinite(temp.y)) temp.y = 0.;

    d_forces[idx]=temp;

    };


__global__ void gpu_force_sets_kernel(float2      *d_points,
                                          int     *d_nn,
                                          float2  *d_AP,
                                          float2  *d_APpref,
                                          int4    *d_delSets,
                                          int     *d_delOther,
                                          float2  *d_forceSets,
                                          float   KA,
                                          float   KP,
                                          int     computations,
                                          int     neighMax,
                                          Index2D n_idx,
                                          gpubox Box
                                        )
    {
    unsigned int tidx = blockDim.x * blockIdx.x + threadIdx.x;
    if (tidx >= computations)
        return;
    //which particle are we evaluating, and which neighbor
    int pidx = tidx / neighMax;
    int nn = tidx - pidx*neighMax;
    //how many neighbors does it have?
    int pNeighbors = d_nn[pidx];

    if(nn >=pNeighbors)
        return;
    //Great...access the four Delaunay neighbors and the relevant fifth point
    float2 pi   = d_points[pidx];

    int4 neighs = d_delSets[n_idx(nn,pidx)];
    float2 pnm2,rij, rik,pn2,pno;
//if(true)    printf("tidx:%i,   pidx:%i,   nm %i,  %i %i %i %i \n",tidx,pidx,neighMax,neighs.x,neighs.y,neighs.z,neighs.w);

    Box.minDist(d_points[neighs.x],pi,pnm2);
    Box.minDist(d_points[neighs.y],pi,rij);
    Box.minDist(d_points[neighs.z],pi,rik);
    Box.minDist(d_points[neighs.w],pi,pn2);
    Box.minDist(d_points[d_delOther[n_idx(nn,pidx)]],pi,pno);

    //first, compute the derivative of the main voro point w/r/t pidx's position
    //pnm1 is rij, pn1 is rik
    Matrix2x2 dhdr;
    Matrix2x2 Id;
    float2 rjk;
    rjk.x =rik.x-rij.x;
    rjk.y =rik.y-rij.y;
    float2 dbDdri,dgDdri,dDdriOD,z;
    float betaD = -dot(rik,rik)*dot(rij,rjk);
    float gammaD = dot(rij,rij)*dot(rik,rjk);
    float cp = rij.x*rjk.y - rij.y*rjk.x;
    float D = 2*cp*cp;
    z.x = betaD*rij.x+gammaD*rik.x;
    z.y = betaD*rij.y+gammaD*rik.y;

    dbDdri.x = 2*dot(rij,rjk)*rik.x+dot(rik,rik)*rjk.x;
    dbDdri.y = 2*dot(rij,rjk)*rik.y+dot(rik,rik)*rjk.y;

    dgDdri.x = -2*dot(rik,rjk)*rij.x-dot(rij,rij)*rjk.x;
    dgDdri.y = -2*dot(rik,rjk)*rij.y-dot(rij,rij)*rjk.y;

    dDdriOD.x = (-2.0*rjk.y)/cp;
    dDdriOD.y = (2.0*rjk.x)/cp;

    dhdr = Id+1.0/D*(dyad(rij,dbDdri)+dyad(rik,dgDdri)-(betaD+gammaD)*Id-dyad(z,dDdriOD));



    //finally, compute all of the forces
    float2 origin; origin.x = 0.0;origin.y=0.0;
    float2 vlast,vcur,vnext,vother;
    Circumcenter(origin,pnm2,rij,vlast);
    Circumcenter(origin,rij,rik,vcur);
    Circumcenter(origin,rik,pn2,vnext);
    Circumcenter(rij,rik,pno,vother);


    float2 dAdv,dPdv;
    float2 dEdv;
    float  Adiff, Pdiff;
    float2 dlast, dnext;
    float  dlnorm,dnnorm;

    //self terms
    dAdv.x = 0.5*(vlast.y-vnext.y);
    dAdv.y = 0.5*(vnext.x-vlast.x);
    dlast.x = vlast.x-vcur.x;
    dlast.y=vlast.y-vcur.y;
    dlnorm = sqrt(dlast.x*dlast.x+dlast.y*dlast.y);
    dnext.x = vcur.x-vnext.x;
    dnext.y = vcur.y-vnext.y;
    dnnorm = sqrt(dnext.x*dnext.x+dnext.y*dnext.y);
    if(dnnorm < THRESHOLD)
        dnnorm = THRESHOLD;
    if(dlnorm < THRESHOLD)
        dlnorm = THRESHOLD;
    dPdv.x = dlast.x/dlnorm - dnext.x/dnnorm;
    dPdv.y = dlast.y/dlnorm - dnext.y/dnnorm;
    Adiff = KA*(d_AP[pidx].x - d_APpref[pidx].x);
    Pdiff = KA*(d_AP[pidx].y - d_APpref[pidx].y);

    dEdv.x  = 2.0*Adiff*dAdv.x +2.0*Pdiff*dPdv.x;
    dEdv.y  = 2.0*Adiff*dAdv.y +2.0*Pdiff*dPdv.y;

    //other terms...k first...
    dAdv.x = 0.5*(vnext.y-vother.y);
    dAdv.y = 0.5*(vother.x-vnext.x);
    dlast.x = vnext.x-vcur.x;
    dlast.y=vnext.y-vcur.y;
    dlnorm = sqrt(dlast.x*dlast.x+dlast.y*dlast.y);
    dnext.x = vcur.x-vother.x;
    dnext.y = vcur.y-vother.y;
    dnnorm = sqrt(dnext.x*dnext.x+dnext.y*dnext.y);
    if(dnnorm < THRESHOLD)
        dnnorm = THRESHOLD;
    if(dlnorm < THRESHOLD)
        dlnorm = THRESHOLD;
    dPdv.x = dlast.x/dlnorm - dnext.x/dnnorm;
    dPdv.y = dlast.y/dlnorm - dnext.y/dnnorm;
    Adiff = KA*(d_AP[neighs.z].x - d_APpref[neighs.z].x);
    Pdiff = KA*(d_AP[neighs.z].y - d_APpref[neighs.z].y);

    dEdv.x  += 2.0*Adiff*dAdv.x +2.0*Pdiff*dPdv.x;
    dEdv.y  += 2.0*Adiff*dAdv.y +2.0*Pdiff*dPdv.y;

    //...and then j
    dAdv.x = 0.5*(vother.y-vlast.y);
    dAdv.y = 0.5*(vlast.x-vother.x);
    dlast.x = vother.x-vcur.x;
    dlast.y=vother.y-vcur.y;
    dlnorm = sqrt(dlast.x*dlast.x+dlast.y*dlast.y);
    dnext.x = vcur.x-vlast.x;
    dnext.y = vcur.y-vlast.y;
    dnnorm = sqrt(dnext.x*dnext.x+dnext.y*dnext.y);
    if(dnnorm < THRESHOLD)
        dnnorm = THRESHOLD;
    if(dlnorm < THRESHOLD)
        dlnorm = THRESHOLD;
    dPdv.x = dlast.x/dlnorm - dnext.x/dnnorm;
    dPdv.y = dlast.y/dlnorm - dnext.y/dnnorm;
    Adiff = KA*(d_AP[neighs.y].x - d_APpref[neighs.y].x);
    Pdiff = KA*(d_AP[neighs.y].y - d_APpref[neighs.y].y);

    dEdv.x  += 2.0*Adiff*dAdv.x +2.0*Pdiff*dPdv.x;
    dEdv.y  += 2.0*Adiff*dAdv.y +2.0*Pdiff*dPdv.y;

    d_forceSets[n_idx(nn,pidx)] = dEdv*dhdr;

//    if(pidx == 0) printf("(%f,%f)\t(%f,%f)\t(%f,%f)\n",dPidv.x,dPidv.y,dPkdv.x,dPkdv.y,dPjdv.x,dPjdv.y);
    //if(pidx == 0) printf("%i %f %f\n",nn,temp.x,temp.y);
//    if(pidx == 0) printf("%f\t%f\t%f\t%f\n",dhdr.x11,dhdr.x12,dhdr.x21,dhdr.x22);

    return;
    };

__global__ void gpu_force_sets_tensions_kernel(float2      *d_points,
                                          int     *d_nn,
                                          float2  *d_AP,
                                          float2  *d_APpref,
                                          int4    *d_delSets,
                                          int     *d_delOther,
                                          float2  *d_forceSets,
                                          int     *d_cellTypes,
                                          float   KA,
                                          float   KP,
                                          float   gamma,
                                          int     computations,
                                          int     neighMax,
                                          Index2D n_idx,
                                          gpubox Box
                                        )
    {
    unsigned int tidx = blockDim.x * blockIdx.x + threadIdx.x;
    if (tidx >= computations)
        return;

    //which particle are we evaluating, and which neighbor
    int pidx = tidx / neighMax;
    int nn = tidx - pidx*neighMax;
    //how many neighbors does it have?
    int pNeighbors = d_nn[pidx];

    if(nn >=pNeighbors)
        return;
    //Great...access the four Delaunay neighbors and the relevant fifth point
    float2 pi   = d_points[pidx];

    int4 neighs = d_delSets[n_idx(nn,pidx)];
    int neighOther = d_delOther[n_idx(nn,pidx)];
    float2 pnm2,rij, rik,pn2,pno;

    Box.minDist(d_points[neighs.x],pi,pnm2);
    Box.minDist(d_points[neighs.y],pi,rij);
    Box.minDist(d_points[neighs.z],pi,rik);
    Box.minDist(d_points[neighs.w],pi,pn2);
    Box.minDist(d_points[neighOther],pi,pno);

    //first, compute the derivative of the main voro point w/r/t pidx's position
    //pnm1 is rij, pn1 is rik
    Matrix2x2 dhdr;
    Matrix2x2 Id;
    float2 rjk;
    rjk.x =rik.x-rij.x;
    rjk.y =rik.y-rij.y;
    float2 dbDdri,dgDdri,dDdriOD,z;
    float betaD = -dot(rik,rik)*dot(rij,rjk);
    float gammaD = dot(rij,rij)*dot(rik,rjk);
    float cp = rij.x*rjk.y - rij.y*rjk.x;
    float D = 2*cp*cp;
    z.x = betaD*rij.x+gammaD*rik.x;
    z.y = betaD*rij.y+gammaD*rik.y;

    dbDdri.x = 2*dot(rij,rjk)*rik.x+dot(rik,rik)*rjk.x;
    dbDdri.y = 2*dot(rij,rjk)*rik.y+dot(rik,rik)*rjk.y;

    dgDdri.x = -2*dot(rik,rjk)*rij.x-dot(rij,rij)*rjk.x;
    dgDdri.y = -2*dot(rik,rjk)*rij.y-dot(rij,rij)*rjk.y;

    dDdriOD.x = (-2.0*rjk.y)/cp;
    dDdriOD.y = (2.0*rjk.x)/cp;

    dhdr = Id+1.0/D*(dyad(rij,dbDdri)+dyad(rik,dgDdri)-(betaD+gammaD)*Id-dyad(z,dDdriOD));



    //finally, compute all of the forces
    float2 origin; origin.x = 0.0;origin.y=0.0;
    float2 vlast,vcur,vnext,vother;
    Circumcenter(origin,pnm2,rij,vlast);
    Circumcenter(origin,rij,rik,vcur);
    Circumcenter(origin,rik,pn2,vnext);
    Circumcenter(rij,rik,pno,vother);


    float2 dAdv,dPdv,dTdv;
    float2 dEdv;
    float  Adiff, Pdiff;
    float2 dlast, dnext;
    float  dlnorm,dnnorm;
    bool Tik = false;
    bool Tij = false;
    bool Tjk = false;
    if (d_cellTypes[pidx] != d_cellTypes[neighs.z]) Tik = true;
    if (d_cellTypes[pidx] != d_cellTypes[neighs.y]) Tij = true;
    if (d_cellTypes[neighs.z] != d_cellTypes[neighs.y]) Tjk = true;
//neighs.z is "baseNeigh" of cpu routing... neighs.y is "otherNeigh"....neighOther is "DT_other_idx"
    //self terms
    dAdv.x = 0.5*(vlast.y-vnext.y);
    dAdv.y = 0.5*(vnext.x-vlast.x);
    dlast.x = vlast.x-vcur.x;
    dlast.y=vlast.y-vcur.y;
    dlnorm = sqrt(dlast.x*dlast.x+dlast.y*dlast.y);
    dnext.x = vcur.x-vnext.x;
    dnext.y = vcur.y-vnext.y;
    dnnorm = sqrt(dnext.x*dnext.x+dnext.y*dnext.y);
    if(dnnorm < THRESHOLD)
        dnnorm = THRESHOLD;
    if(dlnorm < THRESHOLD)
        dlnorm = THRESHOLD;
    dPdv.x = dlast.x/dlnorm - dnext.x/dnnorm;
    dPdv.y = dlast.y/dlnorm - dnext.y/dnnorm;
    dTdv.x = 0.0; dTdv.y = 0.0;
    if(Tik)
        {
        dTdv.x -= dnext.x/dnnorm;
        dTdv.y -= dnext.y/dnnorm;
        };
    if(Tij)
        {
        dTdv.x += dlast.x/dlnorm;
        dTdv.y += dlast.y/dlnorm;
        };

    Adiff = KA*(d_AP[pidx].x - d_APpref[pidx].x);
    Pdiff = KA*(d_AP[pidx].y - d_APpref[pidx].y);

    dEdv.x  = 2.0*Adiff*dAdv.x +2.0*Pdiff*dPdv.x + gamma*dTdv.x;
    dEdv.y  = 2.0*Adiff*dAdv.y +2.0*Pdiff*dPdv.y + gamma*dTdv.y;

    //other terms...k first...
    dAdv.x = 0.5*(vnext.y-vother.y);
    dAdv.y = 0.5*(vother.x-vnext.x);
    dlast.x = vnext.x-vcur.x;
    dlast.y=vnext.y-vcur.y;
    dlnorm = sqrt(dlast.x*dlast.x+dlast.y*dlast.y);
    dnext.x = vcur.x-vother.x;
    dnext.y = vcur.y-vother.y;
    dnnorm = sqrt(dnext.x*dnext.x+dnext.y*dnext.y);
    if(dnnorm < THRESHOLD)
        dnnorm = THRESHOLD;
    if(dlnorm < THRESHOLD)
        dlnorm = THRESHOLD;
    dPdv.x = dlast.x/dlnorm - dnext.x/dnnorm;
    dPdv.y = dlast.y/dlnorm - dnext.y/dnnorm;
    Adiff = KA*(d_AP[neighs.z].x - d_APpref[neighs.z].x);
    Pdiff = KA*(d_AP[neighs.z].y - d_APpref[neighs.z].y);
    dTdv.x = 0.0; dTdv.y = 0.0;
    if(Tik)
        {
        dTdv.x += dlast.x/dlnorm;
        dTdv.y += dlast.y/dlnorm;
        };
    if(Tjk)
        {
        dTdv.x -= dnext.x/dnnorm;
        dTdv.y -= dnext.y/dnnorm;
        };

    dEdv.x  += 2.0*Adiff*dAdv.x +2.0*Pdiff*dPdv.x + gamma*dTdv.x;
    dEdv.y  += 2.0*Adiff*dAdv.y +2.0*Pdiff*dPdv.y + gamma*dTdv.y;

    //...and then j
    dAdv.x = 0.5*(vother.y-vlast.y);
    dAdv.y = 0.5*(vlast.x-vother.x);
    dlast.x = vother.x-vcur.x;
    dlast.y=vother.y-vcur.y;
    dlnorm = sqrt(dlast.x*dlast.x+dlast.y*dlast.y);
    dnext.x = vcur.x-vlast.x;
    dnext.y = vcur.y-vlast.y;
    dnnorm = sqrt(dnext.x*dnext.x+dnext.y*dnext.y);
    if(dnnorm < THRESHOLD)
        dnnorm = THRESHOLD;
    if(dlnorm < THRESHOLD)
        dlnorm = THRESHOLD;
    dPdv.x = dlast.x/dlnorm - dnext.x/dnnorm;
    dPdv.y = dlast.y/dlnorm - dnext.y/dnnorm;
    Adiff = KA*(d_AP[neighs.y].x - d_APpref[neighs.y].x);
    Pdiff = KA*(d_AP[neighs.y].y - d_APpref[neighs.y].y);
    dTdv.x = 0.0; dTdv.y = 0.0;
    if(Tij)
        {
        dTdv.x -= dnext.x/dnnorm;
        dTdv.y -= dnext.y/dnnorm;
        };
    if(Tjk)
        {
        dTdv.x += dlast.x/dlnorm;
        dTdv.y += dlast.y/dlnorm;
        };

    dEdv.x  += 2.0*Adiff*dAdv.x +2.0*Pdiff*dPdv.x + gamma*dTdv.x;
    dEdv.y  += 2.0*Adiff*dAdv.y +2.0*Pdiff*dPdv.y + gamma*dTdv.y;

    d_forceSets[n_idx(nn,pidx)] = dEdv*dhdr;

    return;
    };




__global__ void gpu_compute_geometry_kernel(float2 *d_points,
                                          float2 *d_AP,
                                          float2 *d_voro,
                                          int *d_nn,
                                          int *d_n,
                                          int N,
                                          Index2D n_idx,
                                          gpubox Box
                                        )
    {
    // read in the particle that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;

    float2 circumcenter, origin, nnextp, nlastp,pi,rij,rik,vlast,vnext,vfirst;
    origin.x=0.0;origin.y=0.0;
    int neigh = d_nn[idx];
    float Varea = 0.0;
    float Vperi= 0.0;

    pi = d_points[idx];
    nlastp = d_points[ d_n[n_idx(neigh-1,idx)] ];
    nnextp = d_points[ d_n[n_idx(0,idx)] ];
    Box.minDist(nlastp,pi,rij);
    Box.minDist(nnextp,pi,rik);
    Circumcenter(origin,rij,rik,circumcenter);
    vfirst = circumcenter;
    vlast = circumcenter;
    d_voro[n_idx(0,idx)] = vlast;

    for (int nn = 1; nn < neigh; ++nn)
        {
        rij = rik;
        int nid = d_n[n_idx(nn,idx)];
        nnextp = d_points[ nid ];
        Box.minDist(nnextp,pi,rik);
        Circumcenter(origin,rij,rik,circumcenter);
        vnext = circumcenter;
        d_voro[n_idx(nn,idx)] = circumcenter;

        Varea += TriangleArea(vlast,vnext);
        float dx = vlast.x - vnext.x;
        float dy = vlast.y - vnext.y;
        Vperi += sqrt(dx*dx+dy*dy);
        vlast=vnext;
        };
    Varea += TriangleArea(vlast,vfirst);
    float dx = vlast.x - vfirst.x;
    float dy = vlast.y - vfirst.y;
    Vperi += sqrt(dx*dx+dy*dy);

    d_AP[idx].x=Varea;
    d_AP[idx].y=Vperi;

    return;
    };



__global__ void gpu_displace_and_rotate_kernel(float2 *d_points,
                                          float2 *d_force,
                                          float *d_directors,
                         //                 float2 *d_displacements,
                                          int N,
                                          float dt,
                                          float Dr,
                                          float v0,
                                          int seed,
//                                          hiprandState *states,
                                          gpubox Box
                                         )
    {
    // read in the particle that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;

    hiprandState_t randState;
    hiprand_init(seed*idx,//seed first
                0,   // sequence -- only important for multiple cores
                0,   //offset. advance by sequence by 1 plus this value
                &randState);

    float dirx = cosf(d_directors[idx]);
    float diry = sinf(d_directors[idx]);
    //float angleDiff = hiprand_normal(&states[idx])*sqrt(2.0*dt*Dr);
    float angleDiff = hiprand_normal(&randState)*sqrt(2.0*dt*Dr);
//    printf("%f\n",angleDiff);
    d_directors[idx] += angleDiff;

 //   float dx = dt*(v0*dirx + d_force[idx].x);
//if (idx == 0) printf("x-displacement = %e\n",dx);
//    float f = dt*(v0*dirx + d_force[idx].x);
    d_points[idx].x += dt*(v0*dirx + d_force[idx].x);
//    d_displacements[idx].x = f;

//    f = dt*(v0*diry + d_force[idx].y);
    d_points[idx].y += dt*(v0*diry + d_force[idx].y);
//    d_displacements[idx].y = f;
    Box.putInBoxReal(d_points[idx]);
    return;
    };


//////////////
//kernel callers
//



/*
bool gpu_init_curand(hiprandState *states,
                    unsigned long seed,
                    int N)
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;


    hipMalloc((void **)&states,nblocks*block_size*sizeof(hiprandState) );
    init_curand_kernel<<<nblocks,block_size>>>(seed,states);
    return hipSuccess;
    };
*/


bool gpu_compute_geometry(float2 *d_points,
                        float2   *d_AP,
                        float2   *d_voro,
                        int      *d_nn,
                        int      *d_n,
                        int      N,
                        Index2D  &n_idx,
                        gpubox &Box
                        )
    {
    hipError_t code;
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;

    gpu_compute_geometry_kernel<<<nblocks,block_size>>>(
                                                d_points,
                                                d_AP,
                                                d_voro,
                                                d_nn,
                                                d_n,
                                                N,
                                                n_idx,
                                                Box
                                                );

    code = hipGetLastError();
    if(code!=hipSuccess)
    printf("compute geometry GPUassert: %s \n", hipGetErrorString(code));

    return hipSuccess;
    };


bool gpu_displace_and_rotate(float2 *d_points,
                        float2 *d_force,
                        float  *d_directors,
  //                      float2 *d_displacements,
                        int N,
                        float dt,
                        float Dr,
                        float v0,
                        int seed,
  //                      hiprandState *states,
                        gpubox &Box
                        )
    {
    hipError_t code;
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;

    gpu_displace_and_rotate_kernel<<<nblocks,block_size>>>(
                                                d_points,
                                                d_force,
                                                d_directors,
    //                                            d_displacements,
                                                N,
                                                dt,
                                                Dr,
                                                v0,
                                                seed,
    //                                            states,
                                                Box
                                                );
    code = hipGetLastError();
    if(code!=hipSuccess)
    printf("displaceAndRotate GPUassert: %s \n", hipGetErrorString(code));

    return hipSuccess;
    };

bool gpu_force_sets(float2 *d_points,
                    int    *d_nn,
                    float2 *d_AP,
                    float2 *d_APpref,
                    int4   *d_delSets,
                    int    *d_delOther,
                    float2 *d_forceSets,
                    float2 *d_forces,
                    float  KA,
                    float  KP,
                    int    N,
                    int    neighMax,
                    Index2D &n_idx,
                    gpubox &Box
                    )
    {
    hipError_t code;

    int computations = N*neighMax;
    unsigned int block_size = 128;
    if (computations < 128) block_size = 32;
    unsigned int nblocks  = computations/block_size + 1;

    gpu_force_sets_kernel<<<nblocks,block_size>>>(
                                                d_points,
                                                d_nn,
                                                d_AP,
                                                d_APpref,
                                                d_delSets,
                                                d_delOther,
                                                d_forceSets,
                                                KA,
                                                KP,
                                                computations,
                                                neighMax,
                                                n_idx,
                                                Box
                                                );
    code = hipGetLastError();
    if(code!=hipSuccess)
    printf("forceSets GPUassert: %s \n", hipGetErrorString(code));

    hipDeviceSynchronize();
    //Now sum the forces
    if (computations < 128) block_size = 32;
    nblocks = N/block_size + 1;

    gpu_sum_forces_kernel<<<nblocks,block_size>>>(
                                            d_forceSets,
                                            d_forces,
                                            d_nn,
                                            N,
                                            n_idx
            );

    if(code!=hipSuccess)
    printf("force_sum GPUassert: %s \n", hipGetErrorString(code));


    return hipSuccess;
    };

bool gpu_force_sets_tensions(float2 *d_points,
                    int    *d_nn,
                    float2 *d_AP,
                    float2 *d_APpref,
                    int4   *d_delSets,
                    int    *d_delOther,
                    float2 *d_forceSets,
                    float2 *d_forces,
                    int    *d_cellTypes,
                    float  KA,
                    float  KP,
                    float  gamma,
                    int    N,
                    int    neighMax,
                    Index2D &n_idx,
                    gpubox &Box
                    )
    {
    hipError_t code;

    int computations = N*neighMax;
    unsigned int block_size = 128;
    if (computations < 128) block_size = 32;
    unsigned int nblocks  = computations/block_size + 1;

    gpu_force_sets_tensions_kernel<<<nblocks,block_size>>>(
                                                d_points,
                                                d_nn,
                                                d_AP,
                                                d_APpref,
                                                d_delSets,
                                                d_delOther,
                                                d_forceSets,
                                                d_cellTypes,
                                                KA,
                                                KP,
                                                gamma,
                                                computations,
                                                neighMax,
                                                n_idx,
                                                Box
                                                );
    code = hipGetLastError();
    if(code!=hipSuccess)
    printf("forceSets GPUassert: %s \n", hipGetErrorString(code));

    hipDeviceSynchronize();
    //Now sum the forces
    if (computations < 128) block_size = 32;
    nblocks = N/block_size + 1;

    gpu_sum_forces_kernel<<<nblocks,block_size>>>(
                                            d_forceSets,
                                            d_forces,
                                            d_nn,
                                            N,
                                            n_idx
            );

    if(code!=hipSuccess)
    printf("force_sum GPUassert: %s \n", hipGetErrorString(code));


    return hipSuccess;
    };



#endif
