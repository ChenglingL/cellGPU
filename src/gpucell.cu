#include "hip/hip_runtime.h"
#ifndef __GPUCELL_CU__
#define __GPUCELL_CU__

#define NVCC
#define ENABLE_CUDA

#include <hip/hip_runtime.h>
#include "gpucell.cuh"
#include "indexer.h"
#include "gpubox.h"
#include <iostream>
#include <stdio.h>


__global__ void gpu_compute_cell_list_kernel(float2 *d_pt,
                                              unsigned int *d_cell_sizes,
                                              int *d_idx,
                                              int Np,
                                              unsigned int Nmax,
                                              int xsize,
                                              int ysize,
                                              float boxsize,
                                              gpubox Box,
                                              Index2D ci,
                                              Index2D cli,
                                              int *d_assist
                                              )
    {
    // read in the particle that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= Np)
        return;

    float2 pos = d_pt[idx];

    int ibin = max(0,min(xsize-1,(int)floor(pos.x/boxsize)));
    int jbin = max(0,min(xsize-1,(int)floor(pos.y/boxsize)));
    int bin = ci(ibin,jbin);
    //if (bin > xsize*ysize) printf("(%f,%f) -- (%i,%i) in bin %i out of %i... %f \n",pos.x,pos.y,ibin,jbin,bin,xsize*ysize,boxsize);

    unsigned int offset = atomicAdd(&(d_cell_sizes[bin]), 1);
    //unsigned int offset = d_cell_sizes[bin];
//printf("bin %i  offset %i\n",bin,offset);
    //if (offset <= Nmax+1)
    if (offset <= d_assist[0]+1)
        {
        unsigned int write_pos = min(cli(offset, bin),cli.getNumElements());
        d_idx[write_pos] = idx;
//        atomicAdd(&(d_cell_sizes[bin]),1);
        //d_cell_sizes[bin] += 1;
        }
    else
        {
        //atomicMax(&(*Nmax), offset+1);
        //d_assist[0]=d_assist[0]+1;
        d_assist[0]=offset+1;
        //atomicAdd(&(d_assist[0]),1);
        d_assist[1]=1;
//        printf( "nmax violation ...nmax = %i   offset+1 = %i  \n",Nmax,offset+1);
        //atomicInc(&(*Nmax),1);
        //*Nmax = offset; //atomicMax(Nmax,offset+1);
        };

    return;
    };


bool gpu_compute_cell_list(float2 *d_pt,
                                  unsigned int *d_cell_sizes,
                                  int *d_idx,
                                  int Np,
                                  int &Nmax,
                                  int xsize,
                                  int ysize,
                                  float boxsize,
                                  voroguppy::gpubox &Box,
                                  Index2D &ci,
                                  Index2D &cli,
                                  int *d_assist
                                  )
    {
    //optimize block size later
    unsigned int block_size = 128;
    if (Np < 128) block_size = 16;
    unsigned int nblocks  = Np/block_size + 1;


    unsigned int nmax = (unsigned int) Nmax;
    //cout << "current NMax = " << nmax <<  endl;
    gpu_compute_cell_list_kernel<<<nblocks, block_size>>>(d_pt,
                                                          d_cell_sizes,
                                                          d_idx,
                                                          Np,
                                                          nmax,
                                                          xsize,
                                                          ysize,
                                                          boxsize,
                                                          Box,
                                                          ci,
                                                          cli,
                                                          d_assist
                                                          );

    return hipSuccess;
    }

#endif
