#include "hip/hip_runtime.h"
#define NVCC
#define ENABLE_CUDA

#include <hip/hip_runtime.h>
#include "hiprand/hiprand_kernel.h"
#include "selfPropelledParticleDynamics.cuh"

/** \file selfPropelledParticleDynamics.cu
    * Defines kernel callers and kernels for GPU calculations of simple active 2D cell models
*/

/*!
    \addtogroup selfPropelledParticleDynamicsKernels
    @{
*/

/*!
  Each thread -- most likely corresponding to each cell -- is initialized with a different sequence
  of the same seed of a cudaRNG
*/
__global__ void initialize_sppRNG_kernel(hiprandState *state, int N,int Timestep,int GlobalSeed)
    {
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx >=N)
        return;

    hiprand_init(GlobalSeed,idx,Timestep,&state[idx]);
    return;
    };

//!Call the kernel to initialize a different RNG for each particle
bool gpu_initialize_sppRNG(hiprandState *states,
                    int N,
                    int Timestep,
                    int GlobalSeed)
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;


    initialize_sppRNG_kernel<<<nblocks,block_size>>>(states,N,Timestep,GlobalSeed);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

/*!
Each thread calculates the displacement of an individual cell
*/
__global__ void spp_eom_integration_kernel(Dscalar2 *forces,
                                           Dscalar2 *displacements,
                                           Dscalar2 *motility,
                                           Dscalar *cellDirectors,
                                           hiprandState *RNGs,
                                           int N,
                                           Dscalar deltaT,
                                           int Timestep,
                                           Dscalar mu)
    {
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx >=N)
        return;
    hiprandState_t randState;

    randState=RNGs[idx];
    Dscalar dirx = Cos(cellDirectors[idx]);
    Dscalar diry = Sin(cellDirectors[idx]);
    Dscalar v0 = motility[idx].x;
    Dscalar Dr = motility[idx].y;
    Dscalar angleDiff = cur_norm(&randState)*sqrt(2.0*deltaT*Dr);
    cellDirectors[idx] += angleDiff;

    RNGs[idx] = randState;

    displacements[idx].x = deltaT*(v0*dirx + mu*forces[idx].x);
    displacements[idx].y = deltaT*(v0*diry + mu*forces[idx].y);

    return;
    };

//!get the current timesteps vector of displacements into the displacement vector
bool gpu_spp_eom_integration(
                    Dscalar2 *forces,
                    Dscalar2 *displacements,
                    Dscalar2 *motility,
                    Dscalar *cellDirectors,
                    hiprandState *RNGs,
                    int N,
                    Dscalar deltaT,
                    int Timestep,
                    Dscalar mu)
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;


    spp_eom_integration_kernel<<<nblocks,block_size>>>(
                                forces,displacements,motility,cellDirectors,
                                RNGs,
                                N,deltaT,Timestep,mu);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

/** @} */ //end of group declaration

