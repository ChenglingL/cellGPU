#include "hip/hip_runtime.h"
#ifndef __DTEST_CU__
#define __DTEST_CU__

#define NVCC
#define ENABLE_CUDA
#define EPSILON 1e-12

#include <hip/hip_runtime.h>
#include "gpucell.cuh"
#include "indexer.h"
#include "gpubox.h"
#include "cu_functions.h"
#include <iostream>
#include <stdio.h>
#include "DelaunayCheckGPU.cuh"



__global__ void gpu_test_circumcircles_kernel(bool *d_redo,
                                              int *d_circumcircles,
                                              float2 *d_pt,
                                              unsigned int *d_cell_sizes,
                                              int *d_cell_idx,
                                              int Np,
                                              int xsize,
                                              int ysize,
                                              float boxsize,
                                              gpubox Box,
                                              Index2D ci,
                                              Index2D cli
                                              )
    {
    // read in the particle that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= Np)
        return;
//printf("idx = %i\t",idx);

    //the indices of particles forming the circumcircle
    int i1,i2,i3;
    i1 = d_circumcircles[3*idx];
    i2 = d_circumcircles[3*idx+1];
    i3 = d_circumcircles[3*idx+2];
//if (idx  < 1) printf("%i %i %i check",i1,i2,i3);
    //the vertex we will take to be the origin, and its cell position
    float2 v = d_pt[i1];
    float vz = 0.0;
    int ib,jb;
    ib=floor(v.x/boxsize);
    jb=floor(v.y/boxsize);

    
    
    float2 p1real = d_pt[i2];
    float2 p2real = d_pt[i3];

    float2 pt1,pt2;
    Box.minDist(p1real,v,pt1);
    Box.minDist(p2real,v,pt2);

    //get the circumcircle
    float2 Q;
    float rad;
    Circumcircle(vz,vz,pt1.x,pt1.y,pt2.x,pt2.y,
                    Q.x,Q.y,rad);

    //look through cells for other particles
    bool badParticle = false;
    float2 ptnew,toCenter; 
    int wcheck = ceil(rad/boxsize);
    if(wcheck > xsize/2) wcheck = xsize/2;
if(idx <1)
{
//    printf("(%f,%f), (%f,%f)\n",pt1.x,pt1.y,pt2.x,pt2.y);
//    printf("i1 %i, i2 %i, i3 %i, rad %f, cellsize %f wc = %i\n",i1,i2,i3,rad,boxsize,wcheck);

};
    rad = rad*rad;
    for (int ii = -wcheck; ii <= wcheck; ++ii)
        for (int jj = -wcheck; jj <= wcheck; ++jj)
            {
//if(idx <10) printf("%i\t",jj);
            if(badParticle) continue;

            int cx = (ib+ii);
            if(cx < 0) cx += xsize;
            if(cx >= xsize) cx -= xsize;
            int cy = (jb+jj);
            if(cy < 0) cx += ysize;
            if(cy >= xsize) cx -= ysize;

            int bin = ci(cx,cy);
            for (int pp = 0; pp < d_cell_sizes[bin]; ++pp)
                {
                int newidx = d_cell_idx[cli(pp,bin)];

                float2 pnreal = d_pt[newidx];
                Box.minDist(pnreal,v,ptnew);
                Box.minDist(ptnew,Q,toCenter);
//if(idx <10) printf("%i\t",newidx);
                //if it's in the circumcircle, check that its not one of the three points
                if(toCenter.x*toCenter.x+toCenter.y*toCenter.y < rad)
                    {
                    badParticle = true;
                    if (newidx == i1 || newidx == i2 || newidx ==i3) badParticle = false;
                    };

                };

            };// end loop over cells
    if (badParticle)
        {
printf("badparticle for idxs %i %i %i on threadidx%i\n",i1,i2,i3,idx);
        d_redo[idx] = true;
        d_redo[i1] = true;
        d_redo[i2] = true;
        d_redo[i3] = true;
        };

    return;
    };

bool gpu_test_circumcircles(bool *d_redo,
                                  int *d_ccs,
                                  float2 *d_pt,
                                  unsigned int *d_cell_sizes,
                                  int *d_idx,
                                  int Np,
                                  int xsize,
                                  int ysize,
                                  float boxsize,
                                  gpubox &Box,
                                  Index2D &ci,
                                  Index2D &cli
                                  )
    {

    unsigned int block_size = 128;
    if (Np < 128) block_size = 16;
    unsigned int nblocks  = Np/block_size + 1;

    gpu_test_circumcircles_kernel<<<nblocks,block_size>>>(d_redo,
                                              d_ccs,
                                              d_pt,
                                              d_cell_sizes,
                                              d_idx,
                                              Np,
                                              xsize,
                                              ysize,
                                              boxsize,
                                              Box,
                                              ci,
                                              cli
                                              );
    
    hipDeviceSynchronize();
    cout.flush();
    return hipSuccess;
    };






#endif
