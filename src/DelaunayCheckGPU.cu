#include "hip/hip_runtime.h"
#ifndef __DTEST_CU__
#define __DTEST_CU__

#define NVCC
#define ENABLE_CUDA
#define EPSILON 1e-12

#include <hip/hip_runtime.h>
#include "gpucell.cuh"
#include "indexer.h"
#include "gpubox.h"
#include "cu_functions.h"
#include <iostream>
#include <stdio.h>
#include "DelaunayCheckGPU.cuh"



__global__ void gpu_test_circumcircles_kernel(bool *d_redo,
                                              int *d_circumcircles,
                                              float2 *d_pt,
                                              unsigned int *d_cell_sizes,
                                              int *d_cell_idx,
                                              int Np,
                                              int xsize,
                                              int ysize,
                                              float boxsize,
                                              gpubox Box,
                                              Index2D ci,
                                              Index2D cli
                                              )
    {
    // read in the particle that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= Np)
        return;

    //the indices of particles forming the circumcircle
    int i1,i2,i3;
    i1 = d_circumcircles[3*idx];
    i2 = d_circumcircles[3*idx+1];
    i3 = d_circumcircles[3*idx+2];

    //the vertex we will take to be the origin, and its cell position
    float2 v = d_pt[i1];
    float vz = 0.0;
    int ib,jb;
    ib=floor(v.x/boxsize);
    jb=floor(v.y/boxsize);

    
    
    float2 p1real = d_pt[i2];
    float2 p2real = d_pt[i3];

    float2 pt1,pt2;
    Box.minDist(p1real,v,pt1);
    Box.minDist(p2real,v,pt2);

    //get the circumcircle
    float2 Q;
    float rad;
    Circumcircle(vz,vz,pt1.x,pt1.y,pt2.x,pt2.y,
                    Q.x,Q.y,rad);
    rad = rad*rad;

    //look through cells for other particles
    bool badParticle = false;
    float2 ptnew,toCenter; 
    int wcheck = min((int)ceil(rad/boxsize),(int)xsize/2);
    for (int ii = -wcheck; ii <= wcheck; ++ii)
        for (int jj = -wcheck; jj <= wcheck; ++jj)
            {
            if(badParticle) continue;

            int cx = (ib+ii);
            if(cx < 0) cx += xsize;
            if(cx >= xsize) cx -= xsize;
            int cy = (ib+ii);
            if(cy < 0) cx += ysize;
            if(cy >= xsize) cx -= ysize;

            int bin = ci(cx,cy);
            for (int pp = 0; pp < d_cell_sizes[bin]; ++pp)
                {
                int newidx = d_cell_idx[cli(pp,bin)];

                float2 pnreal = d_pt[newidx];
                Box.minDist(pnreal,v,ptnew);
                Box.minDist(ptnew,Q,toCenter);

                //if it's in the circumcircle, check that its not one of the three points
                if(toCenter.x*toCenter.x+toCenter.y*toCenter.y < rad)
                    {
                    badParticle = true;
                    if (newidx == i1 || newidx == i2 || newidx ==i3) badParticle = false;
                    };

                };

            };// end loop over cells
    if (badParticle)
        {
        d_redo[i1] = true;
        d_redo[i2] = true;
        d_redo[i3] = true;
        };

    return;
    };

bool gpu_test_circumcircles(bool *d_redo,
                                  int *d_ccs,
                                  float2 *d_pt,
                                  unsigned int *d_cell_sizes,
                                  int *d_idx,
                                  int Np,
                                  int xsize,
                                  int ysize,
                                  float boxsize,
                                  gpubox &Box,
                                  Index2D &ci,
                                  Index2D &cli
                                  )
    {

    unsigned int block_size = 128;
    if (Np < 128) block_size = 16;
    unsigned int nblocks  = Np/block_size + 1;


    gpu_test_circumcircles_kernel<<<nblocks,block_size>>>(d_redo,
                                              d_ccs,
                                              d_pt,
                                              d_cell_sizes,
                                              d_idx,
                                              Np,
                                              xsize,
                                              ysize,
                                              boxsize,
                                              Box,
                                              ci,
                                              cli
                                              );

    return hipSuccess;
    };






#endif
