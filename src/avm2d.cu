#include "hip/hip_runtime.h"
#define NVCC
#define ENABLE_CUDA

#include <hip/hip_runtime.h>
#include "hiprand/hiprand_kernel.h"
#include "avm2d.cuh"
#include "lock.h"

/** \file avm.cu
    * Defines kernel callers and kernels for GPU calculations of AVM parts
*/

/*!
    \addtogroup avmKernels
    @{
*/

/*!
  Each thread -- corresponding to each Voronoi cell -- is initialized with a different sequence
  of the same seed of a cudaRNG
*/
__global__ void initialize_curand_kernel(hiprandState *state, int N,int Timestep,int GlobalSeed)
    {
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx >=N)
        return;

    hiprand_init(GlobalSeed,idx,Timestep,&state[idx]);
    return;
    };


/*!
  Since the cells are NOT guaranteed to be convex, the area of the cell must take into account any
  self-intersections. The strategy is the same as in the CPU branch.
  */
__global__ void avm_geometry_kernel(
                                   const Dscalar2* __restrict__ d_vertexPositions,
                                   const int*  __restrict__ d_cellVertexNum,
                                   const int*  __restrict__ d_cellVertices,
                                   const int*  __restrict__ d_vertexCellNeighbors,
                                   Dscalar2*  __restrict__ d_voroCur,
                                   Dscalar4*  __restrict__ d_voroLastNext,
                                   Dscalar2*  __restrict__ d_AreaPerimeter,
                                   int N,
                                   Index2D n_idx,
                                   gpubox Box
                                    )
    {
    // read in the cell index that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    int neighs = d_cellVertexNum[idx];
    //Define the vertices of a cell relative to some (any one ) of its vertices to take care of periodic BCs
    Dscalar2 cellPos = d_vertexPositions[ d_cellVertices[n_idx(neighs-2,idx)]];
    Dscalar2 vlast, vcur,vnext;
    Dscalar Varea = 0.0;
    Dscalar Vperi = 0.0;

    vlast.x = 0.0; vlast.y=0.0;
    int vidx = d_cellVertices[n_idx(neighs-1,idx)];
    Box.minDist(d_vertexPositions[vidx],cellPos,vcur);
    for (int nn = 0; nn < neighs; ++nn)
        {
        //for easy force calculation, save the current, last, and next voronoi vertex position
        //in the approprate spot.
        int forceSetIdx = -1;
        for (int ff = 0; ff < 3; ++ff)
            {
            if(forceSetIdx != -1) continue;
            if(d_vertexCellNeighbors[3*vidx+ff]==idx)
                forceSetIdx = 3*vidx+ff;
            };

        vidx = d_cellVertices[n_idx(nn,idx)];
        Box.minDist(d_vertexPositions[vidx],cellPos,vnext);

        //compute area contribution. It is
        // 0.5 * (vcur.x+vnext.x)*(vnext.y-vcur.y)
        Varea += SignedPolygonAreaPart(vcur,vnext);
        Dscalar dx = vcur.x-vnext.x;
        Dscalar dy = vcur.y-vnext.y;
        Vperi += sqrt(dx*dx+dy*dy);
        //save voronoi positions in a convenient form
        d_voroCur[forceSetIdx] = vcur;
        d_voroLastNext[forceSetIdx] = make_Dscalar4(vlast.x,vlast.y,vnext.x,vnext.y);
        //advance the loop
        vlast = vcur;
        vcur = vnext;
        };
    d_AreaPerimeter[idx].x=Varea;
    d_AreaPerimeter[idx].y=Vperi;
    };

/*!
  The force on a vertex has a contribution from how moving that vertex affects each of the neighboring
cells...compute those force sets
*/
__global__ void avm_force_sets_kernel(
                        int      *d_vertexCellNeighbors,
                        Dscalar2 *d_voroCur,
                        Dscalar4 *d_voroLastNext,
                        Dscalar2 *d_AreaPerimeter,
                        Dscalar2 *d_AreaPerimeterPreferences,
                        Dscalar2 *d_vertexForceSets,
                        int nForceSets,
                        Dscalar KA, Dscalar KP)
    {
    // read in the cell index that belongs to this thread
    unsigned int fsidx = blockDim.x * blockIdx.x + threadIdx.x;
    if (fsidx >= nForceSets)
        return;

    Dscalar2 vlast,vnext;

    int cellIdx = d_vertexCellNeighbors[fsidx];
    Dscalar Adiff = KA*(d_AreaPerimeter[cellIdx].x - d_AreaPerimeterPreferences[cellIdx].x);
    Dscalar Pdiff = KP*(d_AreaPerimeter[cellIdx].y - d_AreaPerimeterPreferences[cellIdx].y);

    //vcur = d_voroCur[fsidx];
    vlast.x = d_voroLastNext[fsidx].x;
    vlast.y = d_voroLastNext[fsidx].y;
    vnext.x = d_voroLastNext[fsidx].z;
    vnext.y = d_voroLastNext[fsidx].w;
    computeForceSetAVM(d_voroCur[fsidx],vlast,vnext,Adiff,Pdiff,d_vertexForceSets[fsidx]);
    };

/*!
  the force on a vertex is decomposable into the force contribution from each of its voronoi
  vertices... add 'em up!
  */
__global__ void avm_sum_force_sets_kernel(
                                    Dscalar2*  d_vertexForceSets,
                                    Dscalar2*  d_vertexForces,
                                    int N)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    Dscalar2 ftemp;
    ftemp.x = 0.0; ftemp.y=0.0;
    for (int ff = 0; ff < 3; ++ff)
        {
        ftemp.x += d_vertexForceSets[3*idx+ff].x;
        ftemp.y += d_vertexForceSets[3*idx+ff].y;
        };
    d_vertexForces[idx] = ftemp;
    };

/*!
  In this version of the active vertex model, the motility of a vertex is a straight average of the
  motility of the three adjacent cells
  */
__global__ void avm_displace_vertices_kernel(
                                        Dscalar2 *d_vertexPositions,
                                        Dscalar2 *d_vertexForces,
                                        Dscalar  *d_cellDirectors,
                                        int      *d_vertexCellNeighbors,
                                        Dscalar  v0,
                                        Dscalar  deltaT,
                                        gpubox   Box,
                                        int      Nvertices)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= Nvertices)
        return;

    //the vertex motility is the average of th motility of the connected cells
    int vn1 = d_vertexCellNeighbors[3*idx];
    int vn2 = d_vertexCellNeighbors[3*idx+1];
    int vn3 = d_vertexCellNeighbors[3*idx+2];
    Dscalar directorx =
            (Cos(d_cellDirectors[vn1])+Cos(d_cellDirectors[vn2])+Cos(d_cellDirectors[vn3]))/3.0;
    Dscalar directory =
            (Sin(d_cellDirectors[vn1])+Sin(d_cellDirectors[vn2])+Sin(d_cellDirectors[vn3]))/3.0;
    //update positions from forces and motility


//    printf("cell %f\t %f\n",deltaT*(v0*directorx), deltaT*d_vertexForces[idx].x);


    d_vertexPositions[idx].x += deltaT*(v0*directorx + d_vertexForces[idx].x);
    d_vertexPositions[idx].y += deltaT*(v0*directory + d_vertexForces[idx].y);
    //make sure the vertices stay in the box
    Box.putInBoxReal(d_vertexPositions[idx]);
    };

/*!
  After the vertices have been moved, the directors of the cells have some noise.
  */
__global__ void avm_rotate_directors_kernel(
                                        Dscalar  *d_cellDirectors,
                                        hiprandState *d_curandRNGs,
                                        Dscalar  Dr,
                                        Dscalar  deltaT,
                                        int      Ncells)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= Ncells)
        return;

    //get the per-cell RNG, rotate the director, return the RNG
    hiprandState_t randState;
    randState=d_curandRNGs[idx];
    d_cellDirectors[idx] += cur_norm(&randState)*sqrt(2.0*deltaT*Dr);
    d_curandRNGs[idx] = randState;
    };

/*!
  There will be severe topology mismatches if a cell is involved in more than one T1 transition
  simultaneously (due to incoherent updates of the cellVertices structure). So, go through the
  current list of edges that are marked to take part in a T1 transition and select one edge per
  cell to be flipped on this trip through the functions.
  */
__global__ void avm_one_T1_per_cell_per_vertex_kernel(
                                        int* __restrict__ d_vertexEdgeFlips,
                                        int* __restrict__ d_vertexEdgeFlipsCurrent,
                                        const int* __restrict__ d_vertexNeighbors,
                                        const int* __restrict__ d_vertexCellNeighbors,
                                        const int* __restrict__ d_cellVertexNum,
                                        const int * __restrict__ d_cellVertices,
                                        int *d_finishedFlippingEdges,
                                        Index2D n_idx,
                                        int Ncells)
    {
    unsigned int cell = blockDim.x * blockIdx.x + threadIdx.x;
    if (cell >= Ncells)
        return;

    //look through every vertex of the cell
    int cneigh = d_cellVertexNum[cell];
    int vertex;
    bool skipRestOfCell = false;
    for (int cc = 0; cc < cneigh; ++cc)
        {
        if (skipRestOfCell) continue;
        vertex = d_cellVertices[n_idx(cc,cell)];
        //what are the other cells attached to this vertex? For correctness, only one cell should
        //own each vertex here. For simplicity, only the lowest-indexed cell gets to do any work.
        if(d_vertexCellNeighbors[3*vertex] < cell ||
               d_vertexCellNeighbors[3*vertex+1] < cell ||
               d_vertexCellNeighbors[3*vertex+2] < cell)
            continue;

        if(d_vertexEdgeFlips[3*vertex] == 1)
            {
            d_vertexEdgeFlipsCurrent[3*vertex] = 1;
            d_vertexEdgeFlips[3*vertex] = 0;
            skipRestOfCell = true;
            };
        if (skipRestOfCell) continue;
        if(d_vertexEdgeFlips[3*vertex+1] == 1)
            {
            d_vertexEdgeFlipsCurrent[3*vertex+1] = 1;
            d_vertexEdgeFlips[3*vertex+1] = 0;
            skipRestOfCell = true;
            };
        if (skipRestOfCell) continue;
        if(d_vertexEdgeFlips[3*vertex+2] == 1)
            {
            d_vertexEdgeFlipsCurrent[3*vertex+2] = 1;
            d_vertexEdgeFlips[3*vertex+2] = 0;
            skipRestOfCell = true;
            };
        };
    if (skipRestOfCell)
        d_finishedFlippingEdges[0] = 1;

    };


/*!
  Run through every pair of vertices (once), see if any T1 transitions should be done,
  and see if the cell-vertex list needs to grow
  */
__global__ void avm_simple_T1_test_kernel(Dscalar2* d_vertexPositions,
                                        int      *d_vertexNeighbors,
                                        int      *d_vertexEdgeFlips,
                                        int      *d_vertexCellNeighbors,
                                        int      *d_cellVertexNum,
                                        gpubox   Box,
                                        Dscalar  T1THRESHOLD,
                                        int      NvTimes3,
                                        int      vertexMax,
                                        int      *d_grow)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= NvTimes3)
        return;
    int vertex1 = idx/3;
    int vertex2 = d_vertexNeighbors[idx];
    Dscalar2 edge;
    if(vertex1 < vertex2)
        {
        Box.minDist(d_vertexPositions[vertex1],d_vertexPositions[vertex2],edge);
        if(norm(edge) < T1THRESHOLD)
            {
            d_vertexEdgeFlips[idx]=1;


            //test the number of neighbors of the cells connected to v1 and v2 to see if the
            //cell list should grow this is kind of slow, and I wish I could optimize it away,
            //or at least not test for it during every time step. The latter seems pretty doable.
            if(d_cellVertexNum[d_vertexCellNeighbors[3*vertex1]] == vertexMax)
                d_grow[0] = 1;
            if(d_cellVertexNum[d_vertexCellNeighbors[3*vertex1+1]] == vertexMax)
                d_grow[0] = 1;
            if(d_cellVertexNum[d_vertexCellNeighbors[3*vertex1+2]] == vertexMax)
                d_grow[0] = 1;
            if(d_cellVertexNum[d_vertexCellNeighbors[3*vertex2]] == vertexMax)
                d_grow[0] = 1;
            if(d_cellVertexNum[d_vertexCellNeighbors[3*vertex2+1]] == vertexMax)
                d_grow[0] = 1;
            if(d_cellVertexNum[d_vertexCellNeighbors[3*vertex2+2]] == vertexMax)
                d_grow[0] = 1;
            }
        else
            d_vertexEdgeFlips[idx]=0;
        }
    else
        d_vertexEdgeFlips[idx] = 0;

    };

/*!
  Flip any edge labeled for re-wiring in the vertexEdgeFlipsCurrent list
  */
__global__ void avm_flip_edges_kernel(int* d_vertexEdgeFlipsCurrent,
                                      Dscalar2 *d_vertexPositions,
                                      int      *d_vertexNeighbors,
                                      int      *d_vertexCellNeighbors,
                                      int      *d_cellVertexNum,
                                      int      *d_cellVertices,
                                      int      *d_finishedFlippingEdges,
                                      Dscalar  T1Threshold,
                                      gpubox   Box,
                                      Index2D  n_idx,
                                      int      NvTimes3)
    {
    if (d_finishedFlippingEdges[0]==0) return;
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    //return if the index is out of bounds or if the edge isn't marked for flipping
    if (idx >= NvTimes3 || d_vertexEdgeFlipsCurrent[idx] == 0)
        return;
    //identify the vertices and reset the flag
    int vertex1 = idx/3;
    int vertex2 = d_vertexNeighbors[idx];
    d_vertexEdgeFlipsCurrent[idx] = 0;

//printf("T1 for vertices %i %i ...\n",vertex1,vertex2);

    //Rotate the vertices in the edge and set them at twice their original distance
    Dscalar2 edge;
    Dscalar2 v1 = d_vertexPositions[vertex1];
    Dscalar2 v2 = d_vertexPositions[vertex2];
    Box.minDist(v1,v2,edge);
    if(norm(edge) < T1Threshold) return;

    //Dscalar2 midpoint;
    //midpoint.x = v2.x + 0.5*edge.x;
    //midpoint.y = v2.y + 0.5*edge.y;

    //v1.x = midpoint.x-edge.y;v1.y = midpoint.y+edge.x;
    //v2.x = midpoint.x+edge.y;v2.y = midpoint.y-edge.x;
    v1.x = v2.x + 0.5*edge.x-edge.y;
    v1.y = v2.y + 0.5*edge.y+edge.x;
    v2.x = v2.x + 0.5*edge.x+edge.y;
    v2.y = v2.y + 0.5*edge.y-edge.x;
    Box.putInBoxReal(v1);
    Box.putInBoxReal(v2);
    d_vertexPositions[vertex1] = v1;
    d_vertexPositions[vertex2] = v2;

    //now, do the gross work of cell and vertex rewiring
    int4 cellSet;cellSet.x=-1;cellSet.y=-1;cellSet.z=-1;cellSet.w=-1;
    //int4 vertexSet;
    int2 vertexSet;
    ///////////////////////////////////////////////////
    //TERRIBLE GPU CODE = COPY THE CPU BRANCH LOGIC....
    ///////////////////////////////////////////////////
    int cell1,cell2,cell3,ctest;
    int vlast, vcur, vnext, cneigh;
    cell1 = d_vertexCellNeighbors[3*vertex1];
    cell2 = d_vertexCellNeighbors[3*vertex1+1];
    cell3 = d_vertexCellNeighbors[3*vertex1+2];
    //cell_l doesn't contain vertex 1, so it is the cell neighbor of vertex 2 we haven't found yet
    for (int ff = 0; ff < 3; ++ff)
        {
        ctest = d_vertexCellNeighbors[3*vertex2+ff];
        if(ctest != cell1 && ctest != cell2 && ctest != cell3)
            cellSet.w=ctest;
        };
    //find vertices "c" and "d"
    cneigh = d_cellVertexNum[cellSet.w];
    vlast = d_cellVertices[ n_idx(cneigh-2,cellSet.w) ];
    vcur = d_cellVertices[ n_idx(cneigh-1,cellSet.w) ];
    for (int cn = 0; cn < cneigh; ++cn)
        {
        vnext = d_cellVertices[n_idx(cn,cell1)];
        if(vcur == vertex2) break;
        vlast = vcur;
        vcur = vnext;
        };

    //classify cell1
    cneigh = d_cellVertexNum[cell1];
    vlast = d_cellVertices[ n_idx(cneigh-2,cell1) ];
    vcur = d_cellVertices[ n_idx(cneigh-1,cell1) ];
    for (int cn = 0; cn < cneigh; ++cn)
        {
        vnext = d_cellVertices[n_idx(cn,cell1)];
        if(vcur == vertex1) break;
        vlast = vcur;
        vcur = vnext;
        };
    if(vlast == vertex2)
        cellSet.x = cell1;
    else if(vnext == vertex2)
        cellSet.z = cell1;
    else
        {
        cellSet.y = cell1;
        };

    //classify cell2
    cneigh = d_cellVertexNum[cell2];
    vlast = d_cellVertices[ n_idx(cneigh-2,cell2) ];
    vcur = d_cellVertices[ n_idx(cneigh-1,cell2) ];
    for (int cn = 0; cn < cneigh; ++cn)
        {
        vnext = d_cellVertices[n_idx(cn,cell2)];
        if(vcur == vertex1) break;
        vlast = vcur;
        vcur = vnext;
        };
    if(vlast == vertex2)
        cellSet.x = cell2;
    else if(vnext == vertex2)
        cellSet.z = cell2;
    else
        {
        cellSet.y = cell2;
        };

    //classify cell3
    cneigh = d_cellVertexNum[cell3];
    vlast = d_cellVertices[ n_idx(cneigh-2,cell3) ];
    vcur = d_cellVertices[ n_idx(cneigh-1,cell3) ];
    for (int cn = 0; cn < cneigh; ++cn)
        {
        vnext = d_cellVertices[n_idx(cn,cell3)];
        if(vcur == vertex1) break;
        vlast = vcur;
        vcur = vnext;
        };
    if(vlast == vertex2)
        cellSet.x = cell3;
    else if(vnext == vertex2)
        cellSet.z = cell3;
    else
        {
        cellSet.y = cell3;
        };

    //get the vertexSet by examining cells j and l
    cneigh = d_cellVertexNum[cellSet.y];
    vlast = d_cellVertices[ n_idx(cneigh-2,cellSet.y) ];
    vcur = d_cellVertices[ n_idx(cneigh-1,cellSet.y) ];
    for (int cn = 0; cn < cneigh; ++cn)
        {
        vnext = d_cellVertices[n_idx(cn,cellSet.y)];
        if(vcur == vertex1) break;
        vlast = vcur;
        vcur = vnext;
        };
    //vertexSet.x=vlast;
    //vertexSet.y=vnext;
    vertexSet.x=vnext;
    cneigh = d_cellVertexNum[cellSet.w];
    vlast = d_cellVertices[ n_idx(cneigh-2,cellSet.w) ];
    vcur = d_cellVertices[ n_idx(cneigh-1,cellSet.w) ];
    for (int cn = 0; cn < cneigh; ++cn)
        {
        vnext = d_cellVertices[n_idx(cn,cellSet.w)];
        if(vcur == vertex2) break;
        vlast = vcur;
        vcur = vnext;
        };
    //vertexSet.w=vlast;
    //vertexSet.z=vnext;
    vertexSet.y=vnext;
    ///////////////////////////////////////////////////
    //END OF FIRST CHUNK OF TERRIBLE CODE...but the nightmare isn't over
    ///////////////////////////////////////////////////

    //re-wire the cells and vertices
    //start with the vertex-vertex and vertex-cell  neighbors
    for (int vert = 0; vert < 3; ++vert)
        {
        //vertex-cell neighbors
        if(d_vertexCellNeighbors[3*vertex1+vert] == cellSet.z)
            d_vertexCellNeighbors[3*vertex1+vert] = cellSet.w;
        if(d_vertexCellNeighbors[3*vertex2+vert] == cellSet.x)
            d_vertexCellNeighbors[3*vertex2+vert] = cellSet.y;
        //vertex-vertex neighbors
        if(d_vertexNeighbors[3*vertexSet.x+vert] == vertex1)
            d_vertexNeighbors[3*vertexSet.x+vert] = vertex2;
        if(d_vertexNeighbors[3*vertexSet.y+vert] == vertex2)
            d_vertexNeighbors[3*vertexSet.y+vert] = vertex1;
        if(d_vertexNeighbors[3*vertex1+vert] == vertexSet.x)
            d_vertexNeighbors[3*vertex1+vert] = vertexSet.y;
        if(d_vertexNeighbors[3*vertex2+vert] == vertexSet.y)
            d_vertexNeighbors[3*vertex2+vert] = vertexSet.x;
        };
    //now rewire the cells
    //cell i loses v2 as a neighbor

    cneigh = d_cellVertexNum[cellSet.x];
    int cidx = 0;
    for (int cc = 0; cc < cneigh-1; ++cc)
        {
        if(d_cellVertices[n_idx(cc,cellSet.x)] == vertex2)
            cidx +=1;
        d_cellVertices[n_idx(cc,cellSet.x)] = d_cellVertices[n_idx(cidx,cellSet.x)];
        cidx +=1;
        };
    d_cellVertexNum[cellSet.x] -= 1;

    //cell j gains v2 in between v1 and b, so step through list backwards and insert
    cneigh = d_cellVertexNum[cellSet.y];
    cidx = cneigh;
    int vLocation = cneigh;
    for (int cc = cneigh-1;cc >=0; --cc)
        {
        int cellIndex = d_cellVertices[n_idx(cc,cellSet.y)];
        if(cellIndex == vertex1)
            {
            vLocation = cidx;
            cidx -= 1;
            };
        d_cellVertices[n_idx(cidx,cellSet.y)] = cellIndex;
        cidx -= 1;
        };
    if(cidx ==0)
        d_cellVertices[n_idx(0,cellSet.y)] = vertex2;
    else
        d_cellVertices[n_idx(vLocation,cellSet.y)] = vertex2;
    d_cellVertexNum[cellSet.y] += 1;

    //cell k loses v1 as a neighbor
    cneigh = d_cellVertexNum[cellSet.z];
    cidx = 0;
    for (int cc = 0; cc < cneigh-1; ++cc)
        {
        if(d_cellVertices[n_idx(cc,cellSet.z)] == vertex1)
            cidx +=1;
        d_cellVertices[n_idx(cc,cellSet.z)] = d_cellVertices[n_idx(cidx,cellSet.z)];
        cidx +=1;
        };
    d_cellVertexNum[cellSet.z] -= 1;

    //cell l gains v1 in between v2 and c...copy the logic of cell j
    cneigh = d_cellVertexNum[cellSet.w];
    cidx = cneigh;
    vLocation = cneigh;
    for (int cc = cneigh-1;cc >=0; --cc)
        {
        int cellIndex = d_cellVertices[n_idx(cc,cellSet.w)];
        if(cellIndex == vertex2)
            {
            vLocation = cidx;
            cidx -= 1;
            };
        d_cellVertices[n_idx(cidx,cellSet.w)] = cellIndex;
        cidx -= 1;
        };
    if(cidx ==0)
        d_cellVertices[n_idx(0,cellSet.w)] = vertex1;
    else
        d_cellVertices[n_idx(vLocation,cellSet.w)] = vertex1;
    d_cellVertexNum[cellSet.w] += 1;

    ///////////////////////////////////////////////////
    //END OF COPIED CODE
    ///////////////////////////////////////////////////
    };


/*!
  This function is being deprecated, but is still useful for calculating, e.g. the mean-squared
displacement of the cells without transferring data back to the hose
*/
__global__ void avm_get_cell_positions_kernel(Dscalar2* d_cellPositions,
                                              Dscalar2* d_vertexPositions,
                                              int    * d_nn,
                                              int    * d_n,
                                              int N,
                                              Index2D n_idx,
                                              gpubox Box)
    {
    // read in the cell index that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;

    Dscalar2 vertex, pos, baseVertex;
    pos.x=0.0;pos.y=0.0;
    baseVertex = d_vertexPositions[ d_n[n_idx(0,idx)] ];
    int neighs = d_nn[idx];
    for (int n = 1; n < neighs; ++n)
        {
        Box.minDist(d_vertexPositions[ d_n[n_idx(n,idx)] ],baseVertex,vertex);
        pos.x += vertex.x;
        pos.y += vertex.y;
        };
    pos.x /= neighs;
    pos.y /= neighs;
    pos.x += baseVertex.x;
    pos.y += baseVertex.y;
    Box.putInBoxReal(pos);
    d_cellPositions[idx] = pos;
    };


//!Call the kernel to initialize a different RNG for each particle
bool gpu_initialize_curand(hiprandState *states,
                    int N,
                    int Timestep,
                    int GlobalSeed)
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;


    initialize_curand_kernel<<<nblocks,block_size>>>(states,N,Timestep,GlobalSeed);
    //hipDeviceSynchronize();
    return hipSuccess;
    };

//!Call the kernel to calculate the area and perimeter of each cell
bool gpu_avm_geometry(
                    Dscalar2 *d_vertexPositions,
                    int      *d_cellVertexNum,
                    int      *d_cellVertices,
                    int      *d_vertexCellNeighbors,
                    Dscalar2 *d_voroCur,
                    Dscalar4 *d_voroLastNext,
                    Dscalar2 *d_AreaPerimeter,
                    int      N,
                    Index2D  &n_idx,
                    gpubox   &Box)
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;


    avm_geometry_kernel<<<nblocks,block_size>>>(d_vertexPositions,
                                                d_cellVertexNum,d_cellVertices,
                                                d_vertexCellNeighbors,d_voroCur,
                                                d_voroLastNext,d_AreaPerimeter,
                                                N, n_idx, Box);
    hipDeviceSynchronize();
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

//!Call the kernel to calculate force sets
bool gpu_avm_force_sets(
                    int      *d_vertexCellNeighbors,
                    Dscalar2 *d_voroCur,
                    Dscalar4 *d_voroLastNext,
                    Dscalar2 *d_AreaPerimeter,
                    Dscalar2 *d_AreaPerimeterPreferences,
                    Dscalar2 *d_vertexForceSets,
                    int nForceSets,
                    Dscalar KA, Dscalar KP)
    {
    unsigned int block_size = 128;
    if (nForceSets < 128) block_size = 32;
    unsigned int nblocks  = nForceSets/block_size + 1;

    avm_force_sets_kernel<<<nblocks,block_size>>>(d_vertexCellNeighbors,d_voroCur,d_voroLastNext,
                                                  d_AreaPerimeter,d_AreaPerimeterPreferences,
                                                  d_vertexForceSets,
                                                  nForceSets,KA,KP);
    hipDeviceSynchronize();
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

//!Call the kernel to sum up the force sets to get net force on each vertex
bool gpu_avm_sum_force_sets(
                    Dscalar2 *d_vertexForceSets,
                    Dscalar2 *d_vertexForces,
                    int      Nvertices)
    {
    unsigned int block_size = 128;
    if (Nvertices < 128) block_size = 32;
    unsigned int nblocks  = Nvertices/block_size + 1;


    avm_sum_force_sets_kernel<<<nblocks,block_size>>>(d_vertexForceSets,d_vertexForces,Nvertices);
    hipDeviceSynchronize();
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };


//!Call the kernel to calculate the area and perimeter of each cell
bool gpu_avm_displace_and_rotate(
                    Dscalar2 *d_vertexPositions,
                    Dscalar2 *d_vertexForces,
                    Dscalar  *d_cellDirectors,
                    int      *d_vertexCellNeighbors,
                    hiprandState *d_curandRNGs,
                    Dscalar  v0,
                    Dscalar  Dr,
                    Dscalar  deltaT,
                    gpubox   &Box,
                    int      Nvertices,
                    int      Ncells)
    {
    unsigned int block_size = 128;
    if (Nvertices < 128) block_size = 32;
    unsigned int nblocks  = Nvertices/block_size + 1;

    //displace vertices
    avm_displace_vertices_kernel<<<nblocks,block_size>>>(d_vertexPositions,d_vertexForces,
                                                         d_cellDirectors,d_vertexCellNeighbors,
                                                         v0,deltaT,Box,Nvertices);
    hipDeviceSynchronize();
    HANDLE_ERROR(hipGetLastError());
    //rotate cell directors
    if (Ncells < 128) block_size = 32;
    nblocks = Ncells/block_size + 1;
    avm_rotate_directors_kernel<<<nblocks,block_size>>>(d_cellDirectors,d_curandRNGs,
                                                        Dr,deltaT,Ncells);
    hipDeviceSynchronize();
    HANDLE_ERROR(hipGetLastError());

    return hipSuccess;
    };


//!Call the kernel to test every edge for a T1 event, see if vertexMax needs to increase
bool gpu_avm_test_edges_for_T1(
                    Dscalar2 *d_vertexPositions,
                    int      *d_vertexNeighbors,
                    int      *d_vertexEdgeFlips,
                    int      *d_vertexCellNeighbors,
                    int      *d_cellVertexNum,
                    int      *d_cellVertices,
                    gpubox   &Box,
                    Dscalar  T1THRESHOLD,
                    int      Nvertices,
                    int      vertexMax,
                    int      *d_grow,
                    Index2D  &n_idx)
    {
    unsigned int block_size = 128;
    int NvTimes3 = Nvertices*3;
    if (NvTimes3 < 128) block_size = 32;
    unsigned int nblocks  = NvTimes3/block_size + 1;

    //test edges
    avm_simple_T1_test_kernel<<<nblocks,block_size>>>(
                                                      d_vertexPositions,d_vertexNeighbors,
                                                      d_vertexEdgeFlips,d_vertexCellNeighbors,
                                                      d_cellVertexNum,
                                                      Box,T1THRESHOLD,
                                                      NvTimes3,vertexMax,d_grow);

    hipDeviceSynchronize();
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

//!Call the kernel to flip at most one edge per cell, write to d_finishedFlippingEdges the current state
bool gpu_avm_flip_edges(
                    int      *d_vertexEdgeFlips,
                    int      *d_vertexEdgeFlipsCurrent,
                    Dscalar2 *d_vertexPositions,
                    int      *d_vertexNeighbors,
                    int      *d_vertexCellNeighbors,
                    int      *d_cellVertexNum,
                    int      *d_cellVertices,
                    int      *d_finishedFlippingEdges,
                    Dscalar  T1Threshold,
                    gpubox   &Box,
                    Index2D  &n_idx,
                    int      Nvertices,
                    int      Ncells)
    {
    unsigned int block_size = 128;

    /*The issue is that if a cell is involved in two edge flips done by different threads, the resulting
    data structure for what vertices belong to cells and what cells border which vertex will be
    inconsistently updated.

    The strategy will be to take the d_vertexEdgeFlips list, put at most one T1 per cell per vertex into the
    d_vertexEdgeFlipsCurrent list (erasing it from the d_vertexEdgeFlips list), and swap the edges specified
    by the "current" list. If d_vertexEdgeFlips is empty, we will set d_finishedFlippingEdges to 1. As long
    as it is != 1, the cpp code will continue calling this gpu_avm_flip_edges function.
    */

    //first select a few edges to flip...
    if(Ncells <128) block_size = 32;
    unsigned int nblocks = Ncells/block_size + 1;
    avm_one_T1_per_cell_per_vertex_kernel<<<nblocks,block_size>>>(
                                                                d_vertexEdgeFlips,
                                                                d_vertexEdgeFlipsCurrent,
                                                                d_vertexNeighbors,
                                                                d_vertexCellNeighbors,
                                                                d_cellVertexNum,
                                                                d_cellVertices,
                                                                d_finishedFlippingEdges,
                                                                n_idx,
                                                                Ncells);
    hipDeviceSynchronize();
    HANDLE_ERROR(hipGetLastError());

    //Now flip 'em
    int NvTimes3 = Nvertices*3;
    if (NvTimes3 < 128) block_size = 32;
    nblocks  = NvTimes3/block_size + 1;

    avm_flip_edges_kernel<<<nblocks,block_size>>>(
                                                  d_vertexEdgeFlipsCurrent,d_vertexPositions,d_vertexNeighbors,
                                                  d_vertexCellNeighbors,d_cellVertexNum,d_cellVertices,
                                                  d_finishedFlippingEdges,
                                                  T1Threshold,Box,
                                                  n_idx,NvTimes3);

    hipDeviceSynchronize();
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };


//!Call the kernel to calculate the position of each cell from the position of its vertices
bool gpu_avm_get_cell_positions(
                    Dscalar2 *d_cellPositions,
                    Dscalar2 *d_vertexPositions,
                    int      *d_cellVertexNum,
                    int      *d_cellVertices,
                    int      N,
                    Index2D  &n_idx,
                    gpubox   &Box)
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;


    avm_get_cell_positions_kernel<<<nblocks,block_size>>>(d_cellPositions,d_vertexPositions,
                                                          d_cellVertexNum,d_cellVertices,
                                                          N, n_idx, Box);
    hipDeviceSynchronize();
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

/** @} */ //end of group declaration
