#include "hip/hip_runtime.h"
#define NVCC
#define ENABLE_CUDA

#include <hip/hip_runtime.h>
#include "hiprand/hiprand_kernel.h"
#include "avm2d.cuh"


//!initialize each thread with a different sequence of the same seed of a cudaRNG
__global__ void initialize_curand_kernel(hiprandState *state, int N,int Timestep,int GlobalSeed)
    {
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx >=N)
        return;

    hiprand_init(GlobalSeed,idx,Timestep,&state[idx]);
    return;
    };


//!compute the voronoi vertices for each cell, along with its area and perimeter
__global__ void avm_geometry_kernel(const Dscalar2* __restrict__ d_p,
                                    const Dscalar2* __restrict__ d_v,
                                    const      int* __restrict__ d_cvn,
                                    const      int* __restrict__ d_cv,
                                    const      int* __restrict__ d_vcn,
                                          Dscalar2*  d_vc,
                                          Dscalar4*  d_vln,
                                          Dscalar2* __restrict__ d_AP,
                                          int N,
                                          Index2D n_idx,
                                          gpubox Box
                                        )
    {
    // read in the cell index that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    int neighs = d_cvn[idx];
    Dscalar2 cellPos = d_p[idx];
    Dscalar2 vlast, vcur,vnext;
    Dscalar Varea = 0.0;
    Dscalar Vperi = 0.0;

    int vidx = d_cv[n_idx(neighs-2,idx)];
    Box.minDist(d_v[vidx],cellPos,vlast);
    vidx = d_cv[n_idx(neighs-1,idx)];
    Box.minDist(d_v[vidx],cellPos,vcur);
if (d_cvn[n_idx(d_cvn[idx]-1, idx)] ==d_cvn[n_idx(d_cvn[idx]-2,idx)])
printf("cell double%i\n",idx);
    for (int nn = 0; nn < neighs; ++nn)
        {
        //for easy force calculation, save the current, last, and next voronoi vertex position in the approprate spot.
        int forceSetIdx = -1;
        for (int ff = 0; ff < 3; ++ff)
            {
           if(d_vcn[3*vidx+ff]==idx)
                forceSetIdx = 3*vidx+ff;
            };

if (forceSetIdx <0 || forceSetIdx >= 6*N || idx == 171)
{
printf("forceSetIdx = %i\t vidx = %i\t nn=%i\n",forceSetIdx,vidx,nn);
printf("cell = %i;  vidx is connected to:",idx);
for (int ff = 0; ff < 3; ++ff)
    printf("%i, ",d_vcn[3*vidx+ff]);
printf("\ncell is connected to:");
int cneigh = d_cvn[idx];
for (int ff = 0; ff < cneigh; ++ff)
    printf("%i, ",d_cvn[n_idx(ff,idx)]);
printf("\n");
};
        vidx = d_cv[n_idx(nn,idx)];
        Box.minDist(d_v[vidx],cellPos,vnext);

        //compute area contribution
        Varea += TriangleArea(vcur,vnext);
        Dscalar dx = vcur.x-vnext.x;
        Dscalar dy = vcur.y-vnext.y;
        Vperi += sqrt(dx*dx+dy*dy);
        //save voronoi positions in a convenient form
        d_vc[forceSetIdx] = vcur;
        d_vln[forceSetIdx] = make_Dscalar4(vlast.x,vlast.y,vnext.x,vnext.y);
        //advance the loop
        vlast = vcur;
        vcur = vnext;
        };
    d_AP[idx].x=Varea;
    d_AP[idx].y=Vperi;
    };

//!compute the force on a vertex due to one of the three cells
__global__ void avm_force_sets_kernel(
                        int      *d_vcn,
                        Dscalar2 *d_vc,
                        Dscalar4 *d_vln,
                        Dscalar2 *d_AP,
                        Dscalar2 *d_APpref,
                        Dscalar2 *d_fs,
                        int nForceSets,
                        Dscalar KA, Dscalar KP)
    {
    // read in the cell index that belongs to this thread
    unsigned int fsidx = blockDim.x * blockIdx.x + threadIdx.x;
    if (fsidx >= nForceSets)
        return;

    Dscalar2 vlast,vnext;

    int cellIdx = d_vcn[fsidx];
    Dscalar Adiff = KA*(d_AP[cellIdx].x - d_APpref[cellIdx].x);
    Dscalar Pdiff = KP*(d_AP[cellIdx].y - d_APpref[cellIdx].y);

    //vcur = d_vc[fsidx];
    vlast.x = d_vln[fsidx].x;
    vlast.y = d_vln[fsidx].y;
    vnext.x = d_vln[fsidx].z;
    vnext.y = d_vln[fsidx].w;
    computeForceSetAVM(d_vc[fsidx],vlast,vnext,Adiff,Pdiff,d_fs[fsidx]);
    };



//!sum up the force sets to get the force on each vertex
__global__ void avm_sum_force_sets_kernel(
                                    const Dscalar2* __restrict__ d_fs,
                                    Dscalar2* __restrict__ d_f,
                                    int N)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    Dscalar2 ftemp;
    ftemp.x = 0.0; ftemp.y=0.0;
    for (int ff = 0; ff < 3; ++ff)
        {
        ftemp.x += d_fs[3*idx+ff].x;
        ftemp.y += d_fs[3*idx+ff].y;
        };
    d_f[idx] = ftemp;
    };

//!sum up the force sets to get the force on each vertex
__global__ void avm_displace_vertices_kernel(
                                        Dscalar2 *d_v,
                                        Dscalar2 *d_f,
                                        Dscalar  *d_cd,
                                        int      *d_vcn,
                                        Dscalar  v0,
                                        Dscalar  deltaT,
                                        gpubox   Box,
                                        int      Nvertices)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= Nvertices)
        return;

    //the vertex motility is the average of th motility of the connected cells
    Dscalar directorx = (Cos(d_cd[d_vcn[3*idx]])+Cos(d_cd[d_vcn[3*idx+1]])+Cos(d_cd[d_vcn[3*idx+2]]))/3.0;
    Dscalar directory = (Sin(d_cd[d_vcn[3*idx]])+Sin(d_cd[d_vcn[3*idx+1]])+Sin(d_cd[d_vcn[3*idx+2]]))/3.0;
    //update positions from forces and motility


//    printf("cell %f\t %f\n",deltaT*(v0*directorx), deltaT*d_f[idx].x);


    d_v[idx].x += deltaT*(v0*directorx + d_f[idx].x);
    d_v[idx].y += deltaT*(v0*directory + d_f[idx].y);
    //make sure the vertices stay in the box
    Box.putInBoxReal(d_v[idx]);
    };

//!sum up the force sets to get the force on each vertex
__global__ void avm_rotate_directors_kernel(
                                        Dscalar  *d_cd,
                                        hiprandState *d_cs,
                                        Dscalar  Dr,
                                        Dscalar  deltaT,
                                        int      Ncells)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= Ncells)
        return;

    //get the per-cell RNG, rotate the director, return the RNG
    hiprandState_t randState;
    randState=d_cs[idx];
    d_cd[idx] += cur_norm(&randState)*sqrt(2.0*deltaT*Dr);
    d_cs[idx] = randState;
    };

/*!
Because operations are performed in parallel, the GPU routine will break if the same vertex
is involved in multiple T1 transitions in the same time step. Defend against that by limiting
the number of flips to one.
*/
__global__ void avm_defend_against_multiple_T1_kernel(
                                        int *d_flip,
                                        int *d_vn,
                                        int Nvertices)
    {
    unsigned int vertex1 = blockDim.x * blockIdx.x + threadIdx.x;
    if (vertex1 >= Nvertices)
        return;
    //if the first vertex-neighbor is to be flipped, prevent any other nearby flips
    if (d_flip[3*vertex1] == 1)
        {
        for (int ff = 0; ff < 3; ++ff)
            {
            int vertex2 = d_vn[3*vertex1+ff];
            for(int f2=0;f2 <3; ++f2)
                d_flip[3*vertex2+f2]=0;
            };
        d_flip[3*vertex1+1] = 0;
        d_flip[3*vertex1+2] = 0;
        };

    //if the second vertex-neighbor is to be flipped, prevent any other flips of the two vertices
    if (d_flip[3*vertex1+1] == 1)
        {
        for (int ff = 0; ff < 3; ++ff)
            {
            int vertex2 = d_vn[3*vertex1+ff];
            for(int f2=0;f2 <3; ++f2)
                d_flip[3*vertex2+f2]=0;
            };
        d_flip[3*vertex1+2] = 0;
        };

    //if the third vertex-neighbor is to be flipped, prevent any other flips of the two vertices
    if (d_flip[3*vertex1+2] == 1)
        {
        for (int ff = 0; ff < 3; ++ff)
            {
            int vertex2 = d_vn[3*vertex1+ff];
            for(int f2=0;f2 <3; ++f2)
                d_flip[3*vertex2+f2]=0;
            };
        };
    };


//!Run through every pair of vertices (once), see if any T1 transitions should be done, and see if the cell-vertex list needs to grow
__global__ void avm_simple_T1_test_kernel(Dscalar2* d_v,
                                        int      *d_vn,
                                        int      *d_vflip,
                                        int      *d_vcn,
                                        int      *d_cvn,
                                        gpubox   Box,
                                        Dscalar  T1THRESHOLD,
                                        int      NvTimes3,
                                        int      vertexMax,
                                        int      *d_grow)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= NvTimes3)
        return;
    int vertex1 = idx/3;
    int vertex2 = d_vn[idx];
    Dscalar2 edge;
    if(vertex1 < vertex2)
        {
        Box.minDist(d_v[vertex1],d_v[vertex2],edge);
        if(norm(edge) < T1THRESHOLD)
            {
            d_vflip[idx]=1;
            //test the number of neighbors of the cells connected to v1 and v2 to see if the cell list should grow
            //this is kind of slow, and I wish I could optimize it away, or at least not test for it during
            //every time step. The latter seems pretty doable.
            if(d_cvn[d_vcn[3*vertex1]] == vertexMax)
                d_grow[0] = 1;
            if(d_cvn[d_vcn[3*vertex1+1]] == vertexMax)
                d_grow[0] = 1;
            if(d_cvn[d_vcn[3*vertex1+2]] == vertexMax)
                d_grow[0] = 1;
            if(d_cvn[d_vcn[3*vertex2]] == vertexMax)
                d_grow[0] = 1;
            if(d_cvn[d_vcn[3*vertex2+1]] == vertexMax)
                d_grow[0] = 1;
            if(d_cvn[d_vcn[3*vertex2+2]] == vertexMax)
                d_grow[0] = 1;
            }
        else
            d_vflip[idx]=0;
        }
    else
        d_vflip[idx] = 0;

    };

//!flip any edge label for re-wiring
__global__ void avm_flip_edges_kernel(int* d_vflip,
                                      Dscalar2 *d_v,
                                      int      *d_vn,
                                      int      *d_vcn,
                                      int      *d_cvn,
                                      int      *d_cv,
                                      gpubox   Box,
                                      Index2D  n_idx,
                                      int      NvTimes3)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    //return if the index is out of bounds or if the edge isn't marked for flipping
    if (idx >= NvTimes3 || d_vflip[idx] == 0)
        return;
    //identify the vertices and reset the flag
    int vertex1 = idx/3;
    int vertex2 = d_vn[idx];
    d_vflip[idx] = 0;

    printf("T1 for vertices %i %i ...\n",vertex1,vertex2);

    //Rotate the vertices in the edge and set them at twice their original distance
    Dscalar2 edge;
    Dscalar2 v1 = d_v[vertex1];
    Dscalar2 v2 = d_v[vertex2];
    Box.minDist(v1,v2,edge);

    Dscalar2 midpoint;
    midpoint.x = v2.x + 0.5*edge.x;
    midpoint.y = v2.y + 0.5*edge.y;

    v1.x = midpoint.x-edge.y;v1.y = midpoint.y+edge.x;
    v2.x = midpoint.x+edge.y;v2.y = midpoint.y-edge.x;
    Box.putInBoxReal(v1);
    Box.putInBoxReal(v2);
    d_v[vertex1] = v1;
    d_v[vertex2] = v2;

    //now, do the gross work of cell and vertex rewiring
    int4 cellSet;cellSet.x=-1;cellSet.y=-1;cellSet.z=-1;cellSet.w=-1;
    int4 vertexSet;
    ///////////////////////////////////////////////////
    //TERRIBLE GPU CODE = COPY THE CPU BRANCH LOGIC....
    ///////////////////////////////////////////////////
    int cell1,cell2,cell3,ctest;
    int vlast, vcur, vnext, cneigh;
    cell1 = d_vcn[3*vertex1];
    cell2 = d_vcn[3*vertex1+1];
    cell3 = d_vcn[3*vertex1+2];
    //cell_l doesn't contain vertex 1, so it is the cell neighbor of vertex 2 we haven't found yet
    for (int ff = 0; ff < 3; ++ff)
        {
        ctest = d_vcn[3*vertex2+ff];
        if(ctest != cell1 && ctest != cell2 && ctest != cell3)
            cellSet.w=ctest;
        };
    //find vertices "c" and "d"
    cneigh = d_cvn[cellSet.w];
    vlast = d_cv[ n_idx(cneigh-2,cellSet.w) ];
    vcur = d_cv[ n_idx(cneigh-1,cellSet.w) ];
    for (int cn = 0; cn < cneigh; ++cn)
        {
        vnext = d_cv[n_idx(cn,cell1)];
        if(vcur == vertex2) break;
        vlast = vcur;
        vcur = vnext;
        };

    //classify cell1
    cneigh = d_cvn[cell1];
    vlast = d_cv[ n_idx(cneigh-2,cell1) ];
    vcur = d_cv[ n_idx(cneigh-1,cell1) ];
    for (int cn = 0; cn < cneigh; ++cn)
        {
        vnext = d_cv[n_idx(cn,cell1)];
        if(vcur == vertex1) break;
        vlast = vcur;
        vcur = vnext;
        };
    if(vlast == vertex2)
        cellSet.x = cell1;
    else if(vnext == vertex2)
        cellSet.z = cell1;
    else
        {
        cellSet.y = cell1;
        };

    //classify cell2
    cneigh = d_cvn[cell2];
    vlast = d_cv[ n_idx(cneigh-2,cell2) ];
    vcur = d_cv[ n_idx(cneigh-1,cell2) ];
    for (int cn = 0; cn < cneigh; ++cn)
        {
        vnext = d_cv[n_idx(cn,cell2)];
        if(vcur == vertex1) break;
        vlast = vcur;
        vcur = vnext;
        };
    if(vlast == vertex2)
        cellSet.x = cell2;
    else if(vnext == vertex2)
        cellSet.z = cell2;
    else
        {
        cellSet.y = cell2;
        };

    //classify cell3
    cneigh = d_cvn[cell3];
    vlast = d_cv[ n_idx(cneigh-2,cell3) ];
    vcur = d_cv[ n_idx(cneigh-1,cell3) ];
    for (int cn = 0; cn < cneigh; ++cn)
        {
        vnext = d_cv[n_idx(cn,cell3)];
        if(vcur == vertex1) break;
        vlast = vcur;
        vcur = vnext;
        };
    if(vlast == vertex2)
        cellSet.x = cell3;
    else if(vnext == vertex2)
        cellSet.z = cell3;
    else
        {
        cellSet.y = cell3;
        };

    //get the vertexSet by examining cells j and l
    cneigh = d_cvn[cellSet.y];
    vlast = d_cv[ n_idx(cneigh-2,cellSet.y) ];
    vcur = d_cv[ n_idx(cneigh-1,cellSet.y) ];
    for (int cn = 0; cn < cneigh; ++cn)
        {
        vnext = d_cv[n_idx(cn,cellSet.y)];
        if(vcur == vertex1) break;
        vlast = vcur;
        vcur = vnext;
        };
    vertexSet.x=vlast;
    vertexSet.y=vnext;
    cneigh = d_cvn[cellSet.w];
    vlast = d_cv[ n_idx(cneigh-2,cellSet.w) ];
    vcur = d_cv[ n_idx(cneigh-1,cellSet.w) ];
    for (int cn = 0; cn < cneigh; ++cn)
        {
        vnext = d_cv[n_idx(cn,cellSet.w)];
        if(vcur == vertex2) break;
        vlast = vcur;
        vcur = vnext;
        };
    vertexSet.w=vlast;
    vertexSet.z=vnext;
    ///////////////////////////////////////////////////
    //END OF FIRST CHUNK OF TERRIBLE CODE...but the nightmare isn't over
    ///////////////////////////////////////////////////

    //re-wire the cells and vertices
    //start with the vertex-vertex and vertex-cell  neighbors
    for (int vert = 0; vert < 3; ++vert)
        {
        //vertex-cell neighbors
        if(d_vcn[3*vertex1+vert] == cellSet.z)
            d_vcn[3*vertex1+vert] = cellSet.w;
        if(d_vcn[3*vertex2+vert] == cellSet.x)
            d_vcn[3*vertex2+vert] = cellSet.y;
        //vertex-vertex neighbors
        if(d_vn[3*vertexSet.y+vert] == vertex1)
            d_vn[3*vertexSet.y+vert] = vertex2;
        if(d_vn[3*vertexSet.z+vert] == vertex2)
            d_vn[3*vertexSet.z+vert] = vertex1;
        if(d_vn[3*vertex1+vert] == vertexSet.y)
            d_vn[3*vertex1+vert] = vertexSet.z;
        if(d_vn[3*vertex2+vert] == vertexSet.z)
            d_vn[3*vertex2+vert] = vertexSet.y;
        };
    //now rewire the cells
    //cell i loses v2 as a neighbor

//    printf("(%i,%i)\t cells: (%i %i %i %i), vertices: (%i,%i,%i,%i)\n",vertex1,vertex2,cellSet.x,cellSet.y,cellSet.z,cellSet.w,vertexSet.x,vertexSet.y,vertexSet.z,vertexSet.w);
/*
if(cellSet.x<0)
    {
    printf("(%i,%i)\t cells: (%i %i %i %i), vertices: (%i,%i,%i,%i)\n",vertex1,vertex2,cellSet.x,cellSet.y,cellSet.z,cellSet.w,vertexSet.x,vertexSet.y,vertexSet.z,vertexSet.w);
    cneigh = d_cvn[d_vcn[3*vertex1]];
    printf("Cell 1, Cellidx %i:",d_vcn[3*vertex1]);
    for (int c1 = 0; c1 < cneigh; ++c1)
        printf("%i\t",d_cv[n_idx(c1,d_vcn[3*vertex1])] );
    printf("\n");
    cneigh = d_cvn[d_vcn[3*vertex1+1]];
    printf("Cell 2, Cellidx %i:",d_vcn[3*vertex1+1]);
    for (int c1 = 0; c1 < cneigh; ++c1)
        printf("%i\t",d_cv[n_idx(c1,d_vcn[3*vertex1+1])] );
    printf("\n");
    cneigh = d_cvn[d_vcn[3*vertex1+2]];
    printf("Cell 3, Cellidx %i:",d_vcn[3*vertex1+2]);
    for (int c1 = 0; c1 < cneigh; ++c1)
        printf("%i\t",d_cv[n_idx(c1,d_vcn[3*vertex1+2])] );
    printf("\n");
    };
*/
    cneigh = d_cvn[cellSet.x];
    int cidx = 0;
    for (int cc = 0; cc < cneigh-1; ++cc)
        {
        if(d_cv[n_idx(cc,cellSet.x)] == vertex2)
            cidx +=1;
        d_cv[n_idx(cc,cellSet.x)] = d_cv[n_idx(cidx,cellSet.x)];
        cidx +=1;
        };
    d_cvn[cellSet.x] -= 1;

    //cell j gains v2 in between v1 and b, so step through list backwards and insert
    cneigh = d_cvn[cellSet.y];
    cidx = cneigh;
    int vLocation = cneigh;
    for (int cc = cneigh-1;cc >=0; --cc)
        {
        int cellIndex = d_cv[n_idx(cc,cellSet.y)];
        if(cellIndex == vertex1)
            {
            vLocation = cidx;
            cidx -= 1;
            };
        d_cv[n_idx(cidx,cellSet.y)] = cellIndex;
        cidx -= 1;
        };
    if(cidx ==0)
        d_cv[n_idx(0,cellSet.y)] = vertex2;
    else
        d_cv[n_idx(vLocation,cellSet.y)] = vertex2;
    d_cvn[cellSet.y] += 1;

    //cell k loses v1 as a neighbor
    cneigh = d_cvn[cellSet.z];
    cidx = 0;
    for (int cc = 0; cc < cneigh-1; ++cc)
        {
        if(d_cv[n_idx(cc,cellSet.z)] == vertex1)
            cidx +=1;
        d_cv[n_idx(cc,cellSet.z)] = d_cv[n_idx(cidx,cellSet.z)];
        cidx +=1;
        };
    d_cvn[cellSet.z] -= 1;

    //cell l gains v1 in between v2 and c...copy the logic of cell j
    cneigh = d_cvn[cellSet.w];
    cidx = cneigh;
    vLocation = cneigh;
    for (int cc = cneigh-1;cc >=0; --cc)
        {
        int cellIndex = d_cv[n_idx(cc,cellSet.w)];
        if(cellIndex == vertex2)
            {
            vLocation = cidx;
            cidx -= 1;
            };
        d_cv[n_idx(cidx,cellSet.w)] = cellIndex;
        cidx -= 1;
        };
    if(cidx ==0)
        d_cv[n_idx(0,cellSet.w)] = vertex1;
    else
        d_cv[n_idx(vLocation,cellSet.w)] = vertex1;
    d_cvn[cellSet.w] += 1;

    ///////////////////////////////////////////////////
    //END OF COPIED CODE
    ///////////////////////////////////////////////////
    };


//!compute the average position of the vertices of each cell, store as the "cell position"
__global__ void avm_get_cell_positions_kernel(Dscalar2* d_p,
                                              Dscalar2* d_v,
                                              int    * d_nn,
                                              int    * d_n,
                                              int N,
                                              Index2D n_idx,
                                              gpubox Box)
    {
    // read in the cell index that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;

    Dscalar2 vertex, pos, baseVertex;
    pos.x=0.0;pos.y=0.0;
    baseVertex = d_v[ d_n[n_idx(0,idx)] ];
    int neighs = d_nn[idx];
    for (int n = 1; n < neighs; ++n)
        {
        Box.minDist(d_v[ d_n[n_idx(n,idx)] ],baseVertex,vertex);
        pos.x += vertex.x;
        pos.y += vertex.y;
        };
    pos.x /= neighs;
    pos.y /= neighs;
    pos.x += baseVertex.x;
    pos.y += baseVertex.y;
    Box.putInBoxReal(pos);
    d_p[idx] = pos;
    };





//!Call the kernel to initialize a different RNG for each particle
bool gpu_initialize_curand(hiprandState *states,
                    int N,
                    int Timestep,
                    int GlobalSeed)
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;


    initialize_curand_kernel<<<nblocks,block_size>>>(states,N,Timestep,GlobalSeed);
    //hipDeviceSynchronize();
    return hipSuccess;
    };

//!Call the kernel to calculate the area and perimeter of each cell
bool gpu_avm_geometry(
                    Dscalar2 *d_p,
                    Dscalar2 *d_v,
                    int      *d_cvn,
                    int      *d_cv,
                    int      *d_vcn,
                    Dscalar2 *d_vc,
                    Dscalar4 *d_vln,
                    Dscalar2 *d_AP,
                    int      N,
                    Index2D  &n_idx,
                    gpubox   &Box)
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;


    avm_geometry_kernel<<<nblocks,block_size>>>(d_p,d_v,d_cvn,d_cv,d_vcn,d_vc,d_vln,d_AP,N, n_idx, Box);
    hipDeviceSynchronize();
    hipError_t code = hipGetLastError();
    if(code!=hipSuccess)
        printf("compute geometry GPUassert: %s \n", hipGetErrorString(code));
    return hipSuccess;
    };

//!Call the kernel to calculate force sets
bool gpu_avm_force_sets(
                    int      *d_vcn,
                    Dscalar2 *d_vc,
                    Dscalar4 *d_vln,
                    Dscalar2 *d_AP,
                    Dscalar2 *d_APpref,
                    Dscalar2 *d_fs,
                    int nForceSets,
                    Dscalar KA, Dscalar KP)
    {
    unsigned int block_size = 128;
    if (nForceSets < 128) block_size = 32;
    unsigned int nblocks  = nForceSets/block_size + 1;

    avm_force_sets_kernel<<<nblocks,block_size>>>(d_vcn,d_vc,d_vln,d_AP,d_APpref,d_fs,nForceSets,KA,KP);
    hipError_t code = hipGetLastError();
    if(code!=hipSuccess)
        printf("compute force sets GPUassert: %s \n", hipGetErrorString(code));
    hipDeviceSynchronize();
    return hipSuccess;
    };

//!Call the kernel to sum up the force sets to get net force on each vertex
bool gpu_avm_sum_force_sets(
                    Dscalar2 *d_fs,
                    Dscalar2 *d_f,
                    int      Nvertices)
    {
    unsigned int block_size = 128;
    if (Nvertices < 128) block_size = 32;
    unsigned int nblocks  = Nvertices/block_size + 1;


    avm_sum_force_sets_kernel<<<nblocks,block_size>>>(d_fs,d_f,Nvertices);
    hipError_t code = hipGetLastError();
    hipDeviceSynchronize();
    if(code!=hipSuccess)
        printf("sum force sets GPUassert: %s \n", hipGetErrorString(code));
    return hipSuccess;
    };


//!Call the kernel to calculate the area and perimeter of each cell
bool gpu_avm_displace_and_rotate(
                    Dscalar2 *d_v,
                    Dscalar2 *d_f,
                    Dscalar  *d_cd,
                    int      *d_vcn,
                    hiprandState *d_cs,
                    Dscalar  v0,
                    Dscalar  Dr,
                    Dscalar  deltaT,
                    gpubox   &Box,
                    int      Nvertices,
                    int      Ncells)
    {
    unsigned int block_size = 128;
    if (Nvertices < 128) block_size = 32;
    unsigned int nblocks  = Nvertices/block_size + 1;

    //displace vertices
    avm_displace_vertices_kernel<<<nblocks,block_size>>>(d_v,d_f,d_cd,d_vcn,v0,deltaT,Box,Nvertices);
    hipDeviceSynchronize();
    //rotate cell directors
    if (Ncells < 128) block_size = 32;
    nblocks = Ncells/block_size + 1;
    avm_rotate_directors_kernel<<<nblocks,block_size>>>(d_cd,d_cs,Dr,deltaT,Ncells);
    hipDeviceSynchronize();
    hipError_t code = hipGetLastError();
    if(code!=hipSuccess)
        printf("displace and rotate GPUassert: %s \n", hipGetErrorString(code));

    return hipSuccess;
    };


//!Call the kernel to test every edge for a T1 event, see if vertexMax needs to increase
bool gpu_avm_test_edges_for_T1(
                    Dscalar2 *d_v,
                    int      *d_vn,
                    int      *d_vflip,
                    int      *d_vcn,
                    int      *d_cvn,
                    gpubox   &Box,
                    Dscalar  T1THRESHOLD,
                    int      Nvertices,
                    int      vertexMax,
                    int      *d_grow)
    {
    unsigned int block_size = 128;
    int NvTimes3 = Nvertices*3;
    if (NvTimes3 < 128) block_size = 32;
    unsigned int nblocks  = NvTimes3/block_size + 1;

    //test edges
    avm_simple_T1_test_kernel<<<nblocks,block_size>>>(
            d_v,d_vn,d_vflip,d_vcn,d_cvn,
            Box,
            T1THRESHOLD,
            NvTimes3,vertexMax,d_grow);

    hipDeviceSynchronize();
    hipError_t code = hipGetLastError();
    if(code!=hipSuccess)
        printf("test for T1 GPUassert: %s \n", hipGetErrorString(code));

    //only allow a vertex to be in one T1 transition in a given time step
    if(Nvertices<128) block_size = 32;
    nblocks = Nvertices/block_size + 1;
    avm_defend_against_multiple_T1_kernel<<<nblocks,block_size>>>(
                                        d_vflip,
                                        d_vn,
                                        Nvertices);
    code = hipGetLastError();
    if(code!=hipSuccess)
        printf("One T1 per vertex per timestep GPUassert: %s \n", hipGetErrorString(code));

    return hipSuccess;
    };

//!Call the kernel to test every edge for a T1 event, see if vertexMax needs to increase
bool gpu_avm_flip_edges(
                    int      *d_vflip,
                    Dscalar2 *d_v,
                    int      *d_vn,
                    int      *d_vcn,
                    int      *d_cvn,
                    int      *d_cv,
                    gpubox   &Box,
                    Index2D  &n_idx,
                    int      Nvertices)
    {
    unsigned int block_size = 128;
    int NvTimes3 = Nvertices*3;
    if (NvTimes3 < 128) block_size = 32;
    unsigned int nblocks  = NvTimes3/block_size + 1;


    //test edges
    avm_flip_edges_kernel<<<nblocks,block_size>>>(
            d_vflip,d_v,d_vn,d_vcn,d_cvn,d_cv,
            Box,n_idx,
            NvTimes3);

    hipDeviceSynchronize();
    hipError_t code = hipGetLastError();
    if(code!=hipSuccess)
        printf("flip edges GPUassert: %s \n", hipGetErrorString(code));
    return hipSuccess;
    };


//!Call the kernel to calculate the position of each cell from the position of its vertices
bool gpu_avm_get_cell_positions(
                    Dscalar2 *d_p,
                    Dscalar2 *d_v,
                    int      *d_nn,
                    int      *d_n,
                    int      N, 
                    Index2D  &n_idx, 
                    gpubox   &Box)
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;


    avm_get_cell_positions_kernel<<<nblocks,block_size>>>(d_p,d_v,d_nn,d_n,N, n_idx, Box);
    hipDeviceSynchronize();
    hipError_t code = hipGetLastError();
    if(code!=hipSuccess)
        {
        printf("get cell positions GPUassert: %s \n", hipGetErrorString(code));
        throw std::exception();
        };
    return hipSuccess;
    };

