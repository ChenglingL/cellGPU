#include "hip/hip_runtime.h"
#define NVCC
#define ENABLE_CUDA

#include <hip/hip_runtime.h>
#include "hiprand/hiprand_kernel.h"
#include "avm2d.cuh"


//!initialize each thread with a different sequence of the same seed of a cudaRNG
__global__ void initialize_curand_kernel(hiprandState *state, int N,int Timestep,int GlobalSeed)
    {
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx >=N)
        return;

    hiprand_init(GlobalSeed,idx,Timestep,&state[idx]);
    return;
    };


//!compute the voronoi vertices for each cell, along with its area and perimeter
__global__ void avm_geometry_kernel(const Dscalar2* __restrict__ d_p,
                                    const Dscalar2* __restrict__ d_v,
                                    const      int* __restrict__ d_nn,
                                    const      int* __restrict__ d_n,
                                    const      int* __restrict__ d_vcn,
                                          Dscalar2*  d_vc,
                                          Dscalar4*  d_vln,
                                          Dscalar2* __restrict__ d_AP,
                                          int N,
                                          Index2D n_idx,
                                          gpubox Box
                                        )
    {
    // read in the cell index that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    int neighs = d_nn[idx];
    Dscalar2 cellPos = d_p[idx];
    Dscalar2 vlast, vcur,vnext;
    Dscalar Varea = 0.0;
    Dscalar Vperi = 0.0;

    int vidx = d_n[n_idx(neighs-2,idx)];
    Box.minDist(d_v[vidx],cellPos,vlast);
    vidx = d_n[n_idx(neighs-1,idx)];
    Box.minDist(d_v[vidx],cellPos,vcur);
    for (int nn = 0; nn < neighs; ++nn)
        {
        //for easy force calculation, save the current, last, and next voronoi vertex position in the approprate spot.
        int forceSetIdx = -1;
        for (int ff = 0; ff < 3; ++ff)
            {
           if(d_vcn[3*vidx+ff]==idx)
                forceSetIdx = 3*vidx+ff;
            };

        vidx = d_n[n_idx(nn,idx)];
        Box.minDist(d_v[vidx],cellPos,vnext);
        
        //compute area contribution
        Varea += TriangleArea(vcur,vnext);
        Dscalar dx = vcur.x-vnext.x;
        Dscalar dy = vcur.y-vnext.y;
        Vperi += sqrt(dx*dx+dy*dy);
        //save voronoi positions in a convenient form
        d_vc[forceSetIdx] = vcur;
        d_vln[forceSetIdx] = make_Dscalar4(vlast.x,vlast.y,vnext.x,vnext.y);
        //advance the loop
        vlast = vcur;
        vcur = vnext;
        };
    d_AP[idx].x=Varea;
    d_AP[idx].y=Vperi;
    };

//!compute the force on a vertex due to one of the three cells
__global__ void avm_force_sets_kernel(
                        int      *d_vcn,
                        Dscalar2 *d_vc,
                        Dscalar4 *d_vln,
                        Dscalar2 *d_AP,
                        Dscalar2 *d_APpref,
                        Dscalar2 *d_fs,
                        int nForceSets,
                        Dscalar KA, Dscalar KP)
    {
    // read in the cell index that belongs to this thread
    unsigned int fsidx = blockDim.x * blockIdx.x + threadIdx.x;
    if (fsidx >= nForceSets)
        return;

    Dscalar2 vlast,vcur,vnext;
    Dscalar2 dlast,dnext;
    Dscalar2 dAdv, dPdv;

    int cellIdx = d_vcn[fsidx];
    Dscalar Adiff = KA*(d_AP[cellIdx].x - d_APpref[cellIdx].x);
    Dscalar Pdiff = KP*(d_AP[cellIdx].y - d_APpref[cellIdx].y);

    vcur = d_vc[fsidx];
    vlast.x = d_vln[fsidx].x;
    vlast.y = d_vln[fsidx].y;
    vnext.x = d_vln[fsidx].z;
    vnext.y = d_vln[fsidx].w;

    dAdv.x = 0.5*(vlast.y-vnext.y);
    dAdv.y = 0.5*(vlast.x-vnext.x);

    dlast.x = vlast.x-vcur.x;
    dlast.y = vlast.y-vcur.y;
    Dscalar dlnorm = sqrt(dlast.x*dlast.x+dlast.y*dlast.y);
    dnext.x = vcur.x-vnext.x;
    dnext.y = vcur.y-vnext.y;
    Dscalar dnnorm = sqrt(dnext.x*dnext.x+dnext.y*dnext.y);
    dPdv.x = dlast.x/dlnorm - dnext.x/dnnorm;
    dPdv.y = dlast.y/dlnorm - dnext.y/dnnorm;

    d_fs[fsidx].x = 2.0*Adiff*dAdv.x + 2.0*Pdiff*dPdv.x;
    d_fs[fsidx].y = 2.0*Adiff*dAdv.y + 2.0*Pdiff*dPdv.y;
    };



//!sum up the force sets to get the force on each vertex
__global__ void avm_sum_force_sets_kernel(
                                    const Dscalar2* __restrict__ d_fs,
                                    Dscalar2* __restrict__ d_f,
                                    int N)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    Dscalar2 ftemp;
    ftemp.x = 0.0; ftemp.y=0.0;
    for (int ff = 0; ff < 3; ++ff)
        {
        ftemp.x += d_fs[3*idx+ff].x;
        ftemp.y += d_fs[3*idx+ff].y;
        };
    d_f[idx] = ftemp;
    };

//!Call the kernel to initialize a different RNG for each particle
bool gpu_initialize_curand(hiprandState *states,
                    int N,
                    int Timestep,
                    int GlobalSeed)
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;


    initialize_curand_kernel<<<nblocks,block_size>>>(states,N,Timestep,GlobalSeed);
    //hipDeviceSynchronize();
    return hipSuccess;
    };

//!Call the kernel to calculate the area and perimeter of each cell
bool gpu_avm_geometry(
                    Dscalar2 *d_p,
                    Dscalar2 *d_v,
                    int      *d_nn,
                    int      *d_n,
                    int      *d_vcn,
                    Dscalar2 *d_vc,
                    Dscalar4 *d_vln,
                    Dscalar2 *d_AP,
                    int      N, 
                    Index2D  &n_idx, 
                    gpubox   &Box)
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;


    avm_geometry_kernel<<<nblocks,block_size>>>(d_p,d_v,d_nn,d_n,d_vcn,d_vc,d_vln,d_AP,N, n_idx, Box);
    //hipDeviceSynchronize();
    return hipSuccess;
    };

bool gpu_avm_force_sets(
                    int      *d_vcn,
                    Dscalar2 *d_vc,
                    Dscalar4 *d_vln,
                    Dscalar2 *d_AP,
                    Dscalar2 *d_APpref,
                    Dscalar2 *d_fs,
                    int nForceSets,
                    Dscalar KA, Dscalar KP)
    {
    unsigned int block_size = 128;
    if (nForceSets < 128) block_size = 32;
    unsigned int nblocks  = nForceSets/block_size + 1;

    avm_force_sets_kernel<<<nblocks,block_size>>>(d_vcn,d_vc,d_vln,d_AP,d_APpref,d_fs,nForceSets,KA,KP);
    //hipDeviceSynchronize();
    return hipSuccess;
    };

bool gpu_avm_sum_force_sets(
                    Dscalar2 *d_fs,
                    Dscalar2 *d_f,
                    int      Nvertices)
    {
    unsigned int block_size = 128;
    if (Nvertices < 128) block_size = 32;
    unsigned int nblocks  = Nvertices/block_size + 1;


    avm_sum_force_sets_kernel<<<nblocks,block_size>>>(d_fs,d_f,Nvertices);
    //hipDeviceSynchronize();
    return hipSuccess;
    };


