#include "hip/hip_runtime.h"
#define NVCC
#define ENABLE_CUDA

#include <hip/hip_runtime.h>
#include "hiprand/hiprand_kernel.h"
#include "selfPropelledParticleDynamics.cuh"

/** \file selfPropelledParticleDynamics.cu
    * Defines kernel callers and kernels for GPU calculations of simple active 2D cell models
*/

/*!
    \addtogroup simpleEquationOfMotionKernels
    @{
*/

/*!
Each thread calculates the displacement of an individual cell
*/
__global__ void spp_eom_integration_kernel(Dscalar2 *forces,
                                           Dscalar2 *displacements,
                                           Dscalar2 *motility,
                                           Dscalar *cellDirectors,
                                           hiprandState *RNGs,
                                           int N,
                                           Dscalar deltaT,
                                           int Timestep,
                                           Dscalar mu)
    {
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx >=N)
        return;
    hiprandState_t randState;

    randState=RNGs[idx];
    Dscalar dirx = Cos(cellDirectors[idx]);
    Dscalar diry = Sin(cellDirectors[idx]);
    Dscalar v0 = motility[idx].x;
    Dscalar Dr = motility[idx].y;
    Dscalar angleDiff = cur_norm(&randState)*sqrt(2.0*deltaT*Dr);
    cellDirectors[idx] += angleDiff;

    RNGs[idx] = randState;

    displacements[idx].x = deltaT*(v0*dirx + mu*forces[idx].x);
    displacements[idx].y = deltaT*(v0*diry + mu*forces[idx].y);

    return;
    };

//!get the current timesteps vector of displacements into the displacement vector
bool gpu_spp_eom_integration(
                    Dscalar2 *forces,
                    Dscalar2 *displacements,
                    Dscalar2 *motility,
                    Dscalar *cellDirectors,
                    hiprandState *RNGs,
                    int N,
                    Dscalar deltaT,
                    int Timestep,
                    Dscalar mu)
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;


    spp_eom_integration_kernel<<<nblocks,block_size>>>(
                                forces,displacements,motility,cellDirectors,
                                RNGs,
                                N,deltaT,Timestep,mu);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

/** @} */ //end of group declaration

